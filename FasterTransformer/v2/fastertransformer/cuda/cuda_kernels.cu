#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "fastertransformer/common.h"

#include "cuda_kernels.h"
#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <climits>
#include <cfloat>
namespace fastertransformer{

#define FINAL_MASK 0xffffffff
#define HIP_PI_F 3.141592654f

template <typename T>
__inline__ __device__
T gelu(T x)
{
  float cdf = 0.5f * (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

template <>
__inline__ __device__
half2 gelu(half2 val)
{
  half2 val_pow3 = __hmul2(val, __hmul2(val, val));
  float2 tmp_pow = __half22float2(val_pow3);
  float2 tmp =  __half22float2(val);

  tmp.x = 0.5f * (1.0f + tanhf((0.7978845608028654f * (tmp.x + 0.044715f * tmp_pow.x))));
  tmp.y = 0.5f * (1.0f + tanhf((0.7978845608028654f * (tmp.y + 0.044715f * tmp_pow.y))));
  return __hmul2(val, __float22half2_rn(tmp));
}

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

template <typename T>
__inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;
  __syncthreads();
  
  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)0.0f;
  val = warpReduceSum(val);
  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)-1e20f;
  val = warpReduceMax<T>(val);

  return val;
}


template <typename T>
__global__ 
void add_bias_act(T* out, const T* bias, int m, int n)
{
  T val, reg_bias;

  int row_id = blockIdx.x;
  int ite = n / blockDim.x;
  int tid = threadIdx.x;

  for(int i = 0; i < ite; ++i)
  {
    reg_bias = __ldg(&bias[i * blockDim.x + tid]);
    row_id = blockIdx.x;

    while(row_id < m){
      val = out[tid + i * blockDim.x + row_id * n]+ reg_bias;
      out[tid + i * blockDim.x + row_id * n] = gelu<T>(val);
      row_id += gridDim.x;
    }
  }
}

template <>
__global__ 
void add_bias_act(half* out, const half* bias, int m, int n)
{
  half2 val, reg_bias;
  int row_id = blockIdx.x;
  int ite = n / blockDim.x / 2;
  int tid = threadIdx.x;

  half2* out_ptr = (half2*) out;
  const half2* bias_ptr = (half2*) bias;
  for(int i = 0; i < ite; ++i)
  {
    reg_bias = __ldg(&bias_ptr[i * blockDim.x + tid]);
    row_id = blockIdx.x;

    while(row_id < m){
      val = out_ptr[tid + i * blockDim.x + row_id * n / 2];
      val = __hadd2(val, reg_bias);
      out_ptr[tid + i * blockDim.x + row_id * n / 2] = gelu<half2>(val);
      row_id += gridDim.x;
    }
  }
}

template <typename T>
__global__ 
void add_bias_input_layernorm(T* out, const T* input, const T* bias, const T* gamma, const T* beta, int m, int n)
{
  int tid = threadIdx.x;

  __shared__ float s_mean;
  __shared__ float s_variance;
  float mean =  0.0f;
  float variance = 0.0f;

  float local_out = 0.0f;
  for(int i = tid; i < n; i += blockDim.x)
    local_out += (float)(out[blockIdx.x * n + i] + input[blockIdx.x * n + i] + __ldg(&bias[i]));

  mean = blockReduceSum<float>(local_out);
  if(threadIdx.x == 0)
    s_mean = mean / n;
  __syncthreads();

  variance = blockReduceSum<float>((local_out - s_mean) * (local_out - s_mean));
  if(threadIdx.x == 0)
    s_variance = variance / n + 1e-6f;
  __syncthreads();

  for(int i = tid; i < n; i += blockDim.x)
    out[blockIdx.x * n + i] = 
	    (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(__ldg(&gamma[i])) + (float)(__ldg(&beta[i])));
}

template <>
__global__ 
void add_bias_input_layernorm(half* out, const half* input, const half* bias, 
  const half* gamma, const half* beta, int m, int n)
{

  int tid = threadIdx.x;
  __shared__ float s_mean;
  __shared__ float s_variance;
  float mean =  0.0f;
  float variance = 0.0f;
  float2 local_out_fp2;

  half2* out_ptr = (half2*)out;
  const half2* input_ptr = (const half2*)input;
  const half2* bias_ptr = (const half2*)bias;
  const half2* gamma_ptr = (const half2*)gamma;
  const half2* beta_ptr = (const half2*)beta;
 
  float local_out = 0.0f;
  int id = blockIdx.x * n / 2 + tid; 
  local_out_fp2 = __half22float2(__hadd2(__hadd2(out_ptr[id], input_ptr[id]), __ldg(&bias_ptr[tid])));
  local_out += local_out_fp2.x;
  local_out += local_out_fp2.y;

  mean = blockReduceSum<float>(local_out);
  if(threadIdx.x == 0)
    s_mean = mean / n;
  __syncthreads();

  variance = (local_out_fp2.x - s_mean) * (local_out_fp2.x - s_mean);
  variance += (local_out_fp2.y - s_mean) * (local_out_fp2.y - s_mean);
  variance = blockReduceSum<float>(variance);
  if(threadIdx.x == 0)
    s_variance = rsqrtf(variance / n + 1e-6f);
  __syncthreads();

  float2 gamma_val = __half22float2(__ldg(&gamma_ptr[tid]));
  float2 beta_val = __half22float2(__ldg(&beta_ptr[tid]));
  local_out_fp2.x = (local_out_fp2.x - s_mean) * s_variance * gamma_val.x + beta_val.x;
  local_out_fp2.y = (local_out_fp2.y - s_mean) * s_variance * gamma_val.y + beta_val.y;
  out_ptr[id] = __float22half2_rn(local_out_fp2);
}

template<typename T>
__global__
void broadcast_kernel(T* log_probs, T* cum_log_probs, const int batch_size, const int beam_width, const int vocab_size, const int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int bid = tid / vocab_size;

  if(tid < N)
    log_probs[tid] += cum_log_probs[bid];
}

template <typename T>
void add_bias_act_kernelLauncher(T* out, const T* bias, int m, int n, hipStream_t stream)
{
//  dim3 grid(m / 64);
  dim3 grid(m / 4);
  dim3 block(n / 4);
  assert(block.x <= 1024);
//  dim3 block(n);
  add_bias_act<T><<<grid, block, 0, stream>>>(out, bias, m, n);
}

template<typename T>
void add_bias_input_layernorm_kernelLauncher(T* out, const T* input, const T* bias, 
  const T* gamma, const T* beta, int m, int n, hipStream_t stream)
{
  dim3 grid(m);
  dim3 block(n);
  assert(n <= 1024);
  add_bias_input_layernorm<T><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
}


template <>
void add_bias_input_layernorm_kernelLauncher(half* out, const half* input, const half* bias, 
  const half* gamma, const half* beta, int m, int n, hipStream_t stream)
{
  dim3 grid(m);
  dim3 block(n / 2);
  assert(n / 2 <= 1024);
  add_bias_input_layernorm<half><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
}

void broadcast_kernelLauncher(float* log_probs, float* cum_log_probs, const int batch_size, const int beam_width, 
  const int vocab_size, hipStream_t stream)
{
  
  int N = batch_size * beam_width * vocab_size;
  dim3 block(1024);
  dim3 grid((N - 1) / block.x + 1);

  broadcast_kernel<float><<<grid, block, 0, stream>>>(log_probs, cum_log_probs, batch_size, beam_width, vocab_size, N);
}

template <typename T>
__global__
void topK_kernel(const T* log_probs, int* ids, const int batch_size, const int N, const int K)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float val, max_val;
  __shared__ float s_max_val;
  for(int ite = 0; ite < batch_size; ++ite)
  {
    bool choosed = false;
    val = (tid < N ) ? (float)log_probs[ite * N + tid] : -1e20f;
    
    for(int kids = 0; kids < K; ++kids)
    {
      max_val = blockReduceMax<float>(val);
      
      if(threadIdx.x == 0)
        s_max_val = max_val;
      __syncthreads();

      if(s_max_val == val && !choosed && tid < N) 
      {
        ids[ite * gridDim.x * K + blockIdx.x * K + kids] = tid + ite * N;
        val = -1e20f;
        choosed = true;
      }
    }
  }
}

template <typename T>
__global__
void topK_kernel_2nd(const T* log_probs, int* ids, const int batch_size, const int N, const int K, const int id_offset)
{
  int tid = threadIdx.x;
  float val, max_val;
  __shared__ float s_max_val;
  __shared__ int beam_index;
  __shared__ int ids_before_sort[16];

  for(int ite = 0; ite < batch_size; ++ite)
  {
    bool choosed = false;
    const int id = (tid < N) ? ids[ite * N + tid] : -1;
    val = (tid < N) ? (float)log_probs[id] : -1e20f;

    __syncthreads();

    if(tid == 0) beam_index = 0;
    if(tid < 16) ids_before_sort[tid] = -1;
    
    __syncthreads();
    while(beam_index < K){
      int begin_beam_index = beam_index;
      max_val = blockReduceMax<float>(val);
      if(threadIdx.x == 0){
        s_max_val = max_val;
      }
      __syncthreads();
      if(s_max_val == val && !choosed && id != -1)
      {
        int id_offset_ = atomicAdd(&beam_index, 1);
        ids_before_sort[id_offset_] = id;
        val = -1e20f;
        choosed = true;
      }
      __syncthreads();
     
      // simply sort the ids
      if(threadIdx.x == 0 && beam_index - begin_beam_index > 1){
        for(int i = begin_beam_index; i < beam_index; i++){
          for(int j = i; j < beam_index; j++){
            if(ids_before_sort[j] < ids_before_sort[i]){
              int tmpid = ids_before_sort[j];
              ids_before_sort[j] = ids_before_sort[i];
              ids_before_sort[i] = tmpid;
            }
          }
        }
      }
    }
    __syncthreads();
    if(tid < K) ids[ite * K + tid] = ids_before_sort[tid];
    __syncthreads();
  }
}

void topK(const float* log_probs, int* ids, const int batch_size, const int beam_width, const int vocab_size,
  hipStream_t stream)
{
  int N = beam_width * vocab_size;
  dim3 block(1024);
  dim3 grid((N - 1) / block.x + 1);
  /* First round topK, for each batch, get grid.x * K values */
  topK_kernel<float><<<grid, block, 0, stream>>>(log_probs, ids, batch_size, N, beam_width);
  /*Second round, for each batch, get the final TopK values out from grid.x * K values. */
  topK_kernel_2nd<float><<<1, block, 0, stream>>>(log_probs, ids, batch_size, beam_width * grid.x, beam_width, N);
}

template <typename T>
__global__
void update_kernel(T* log_probs, T* cum_log_probs, 
                  int* ids, bool* finished, 
                  int* parent_ids, int* sequence_length, 
                  int* word_ids, int* output_ids, 
                  const int batch_size, const int beam_width, 
                  const int vocab_size, const int end_id, 
                  int* finished_count)
{
  int tid = threadIdx.x;
  sequence_length[tid] = finished[tid] ? sequence_length[tid] : sequence_length[tid] + 1;

  int beam_id = ids[tid];
  beam_id /= vocab_size;
  int word_id = ids[tid];
  word_id %= vocab_size;

  cum_log_probs[tid] = log_probs[ids[tid]];
  sequence_length[tid] = sequence_length[beam_id];
  finished[tid] = word_id == end_id ? 1 : 0;
  parent_ids[tid] = beam_id;
  word_ids[tid] = word_id;
  output_ids[tid] = word_id;

  // TODO use reduce sum to compute how many sentence are finished
  // int fi = finished[tid]
  // int total_finish = reduceSum(fi);
}

template <typename T>
__global__ void embedding_lookup_kernel(const T* embedding_table, const int* word_ids,
    const int hidden_units, T* from_tensor)
{
  int write_pos = threadIdx.x + blockIdx.x * hidden_units;
  from_tensor[write_pos] = embedding_table[word_ids[blockIdx.x] * hidden_units + threadIdx.x];
}

void update(float* log_probs, float* cum_log_probs, 
            int* ids, bool* finished, 
            int* parent_ids, int* sequence_length,
            int* word_ids, int* output_ids, 
            const int batch_size, const int beam_width, 
            const int vocab_size, hipStream_t stream, 
            const int end_id, int* finished_count)
{ 
  
  dim3 grid(1);
  dim3 block(batch_size * beam_width);

  assert(block.x <= 1024);

  update_kernel<float><<<grid, block, 0, stream>>>(log_probs, cum_log_probs, ids, 
                                                  finished, parent_ids, sequence_length,
                                                  word_ids, output_ids, batch_size, 
                                                  beam_width, vocab_size, end_id, 
                                                  finished_count);
}

template <typename T>
void embedding_lookup(const T* embedding_table, const int* word_ids, T* from_tensor,
  const int batch_size, const int beam_width, const int hidden_units, hipStream_t stream)
{
   dim3 grid(batch_size * beam_width);
   dim3 block(hidden_units);
   assert(hidden_units <= 1024);
   embedding_lookup_kernel<<<grid, block, 0, stream>>>(embedding_table, word_ids, hidden_units, from_tensor);
}

template <typename T>
__global__ void update_logits_kernel(T* logits, const T* bias, const int end_id, const bool* finished, const int n)
{
  int bid = blockIdx.x;
  bool finish = finished[bid];
  int offset = bid * n;

  float max_val = -1 * FLT_MAX;
  __shared__ float s_max_val;
  __shared__ float s_sum_val;

  for(int tid = threadIdx.x; tid < n; tid += blockDim.x)
  {
    if(finish)
      logits[offset + tid] = (tid == end_id) ? FLT_MAX : -1 * FLT_MAX;
    else
      logits[offset + tid] += bias[tid];
    max_val = max(max_val, logits[offset + tid]);
  }

  max_val = blockReduceMax<float>((float)max_val);
  if(threadIdx.x == 0)
    s_max_val = max_val;
  __syncthreads();

  float sum_val = 0.0f;
  for(int tid = threadIdx.x; tid < n; tid += blockDim.x)
  {
    logits[offset + tid] = __expf((float)logits[offset + tid] - s_max_val);
    sum_val += (float)logits[offset + tid];
  }

  sum_val = blockReduceSum<float>(sum_val);
  if(threadIdx.x == 0)
    s_sum_val = sum_val;
  __syncthreads();

  for(int tid = threadIdx.x; tid < n; tid += blockDim.x)
  {
    logits[offset + tid] = logf((float)logits[offset + tid] / s_sum_val);
  }
}

void update_logits(float* logits, const float* bias, const int end_id, const bool* finished, 
  const int m, const int n, hipStream_t stream)
{
  dim3 grid(m);
  dim3 block(min(n, 1024));
  /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
  update_logits_kernel<float><<<grid, block, 0, stream>>>(logits, bias, end_id, finished, n);
}

template <typename T>
__global__ void init_kernel(bool* finished, int* sequence_length, int* word_ids, T* cum_log_probs, const int sentence_id, const int n, const int beam_width)
{
  int tid = threadIdx.x;
  finished[tid] = false;
  sequence_length[tid] = 0;
  word_ids[tid] = sentence_id;
  cum_log_probs[tid] = (T)(tid % beam_width == 0 ? 0.0f: -1e20f);
}

template <typename T>
__global__ void update_KV_cache_kernel(
  T* key_src_cache, T* key_tgt_cache,
  T* value_src_cache, T* value_tgt_cache,
  const int* beam_ids, const int batch_size, const int beam_width, const int hidden_dim, const int cache_size, const int step, const int decoder_layers)
{
  int layer_id = blockIdx.x / batch_size / beam_width / step;
  int batch_id = (blockIdx.x % (batch_size * beam_width * step)) / (beam_width * step);
  int beam_id = (blockIdx.x % (beam_width * step)) / step;
  int step_id = blockIdx.x % step;

  int hidden_id = step_id * batch_size * beam_width * hidden_dim + 
    beam_ids[batch_id * beam_width + beam_id] * hidden_dim;

  int tgt_hidden_id = step_id * batch_size * beam_width * hidden_dim + 
    batch_id * beam_width * hidden_dim + beam_id * hidden_dim;

  T* key_src_ptr = key_src_cache + layer_id * cache_size;
  T* key_tgt_ptr = key_tgt_cache + layer_id * cache_size;
  T* value_src_ptr = value_src_cache + layer_id * cache_size;
  T* value_tgt_ptr = value_tgt_cache + layer_id * cache_size;


  for(int tid = threadIdx.x; tid < hidden_dim; tid += blockDim.x)
  {
    key_tgt_ptr[tgt_hidden_id + tid] = key_src_ptr[hidden_id + tid];
    value_tgt_ptr[tgt_hidden_id + tid] = value_src_ptr[hidden_id + tid];
  }
  
}
template <typename T>
void update_KV_cache(T** key_cache, T** value_cache, const int* beam_ids, const int batch_size, const int beam_width, const int hidden_dim,
  const int step, const int cache_size, const int decoder_layers, hipStream_t stream)
{
  dim3 grid(decoder_layers * batch_size * beam_width * step);
  dim3 block(min(1024, hidden_dim));

  int src_id = step & 0x1;
  int tgt_id = 1 - src_id;

  update_KV_cache_kernel<<<grid, block, 0, stream>>>(
    key_cache[src_id], key_cache[tgt_id],
    value_cache[src_id], value_cache[tgt_id],
    beam_ids, batch_size, beam_width, hidden_dim, cache_size, step, decoder_layers);
}

void init(bool* finished, int* sequence_length, int* word_ids, float* cum_log_probs, const int sentence_id, const int batch_size, 
  const int beam_width, hipStream_t stream)
{
  dim3 grid(1);
  dim3 block(min(1024, batch_size * beam_width));

  assert(batch_size * beam_width <= 1024);
  
  init_kernel<float><<<grid, block, 0, stream>>>(finished, sequence_length, word_ids, cum_log_probs, sentence_id, batch_size * beam_width, beam_width);
}


template<typename T>
__global__
void sine_position_encoder_kernel(T* output, int step, int n){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  float half_n = (float)n / 2.;

  // input = input * hidden_dim**0.5
  output[bid * n + tid] = output[bid * n + tid] * (T)sqrtf(float(n));

  float log_timescale_increment = __logf(10000) / (half_n - 1.f);
  float inv_timescales = __expf( (tid % (int)half_n) * -1 * log_timescale_increment );
  float scaled_time = inv_timescales * step;
  
  T encoding_val = (tid < half_n) ? (T) __sinf(scaled_time) : (T) __cosf(scaled_time);
  output[bid * n + tid] = output[bid * n + tid]  + encoding_val;
}

template<typename T>
void sine_position_encoder(
  T* output,
  int step,
  int m, int n, hipStream_t stream)
{
  dim3 grid(m);
  dim3 block(n);
  assert(n <= 1024);
  sine_position_encoder_kernel<T><<<grid, block, 0, stream>>>(output, step, n);
}

template void add_bias_act_kernelLauncher<float>(
  float* out, const float* bias, int m, int n, hipStream_t stream);

template void add_bias_input_layernorm_kernelLauncher<float>(
  float* out, const float* input, const float* bias, const float* gamma, const float* beta, 
  int m, int n, hipStream_t stream);

template void add_bias_act_kernelLauncher<half>(
  half* out, const half* bias, int m, int n, hipStream_t stream);

template void add_bias_input_layernorm_kernelLauncher<half>(
  half* out, const half* input, const half* bias, const half* gamma, const half* beta, 
  int m, int n, hipStream_t stream);

template void embedding_lookup(const float* embedding_table, const int* word_ids, float* from_tensor,
  const int batch_size, const int beam_width, const int hidden_units, hipStream_t stream);

template void embedding_lookup(const half* embedding_table, const int* word_ids, half* from_tensor,
  const int batch_size, const int beam_width, const int hidden_units, hipStream_t stream);

template void update_KV_cache(float** key_cache, float** value_cache, const int* beam_ids, const int batch_size, const int beam_width, const int hidden_dim,
  const int step, const int cache_size, const int decoder_layers, hipStream_t stream);

template void update_KV_cache(half** key_cache, half** value_cache, const int* beam_ids, const int batch_size, const int beam_width, const int hidden_dim,
  const int step, const int cache_size, const int decoder_layers, hipStream_t stream);

template void sine_position_encoder(
  float* output,
  int step,
  int m, int n,
  hipStream_t stream);

template void sine_position_encoder(
  half* output,
  int step,
  int m, int n,
  hipStream_t stream);

}//namespace 
