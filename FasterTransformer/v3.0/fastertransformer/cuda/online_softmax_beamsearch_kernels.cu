#include "hip/hip_runtime.h"
/*
* Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "fastertransformer/cuda/topk_kernels.cuh"
#include "hipcub/hipcub.hpp"

namespace fastertransformer
{

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE)
__global__
void batch_topK_kernel(const int* __restrict topk_tmp_id_buf,
                        const T* __restrict topk_tmp_val_buf,
                        int* __restrict id_buf,
                        T* __restrict val_buf)
{
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    TopK<T, MAX_K> partial;
    if (thread_id == 0)
    {
        for(int i = 0; i < MAX_K; ++i)
        {
            partial.p[i] = -1;
            partial.u[i] = -FLT_MAX;
        }

        int index = block_id * MAX_K * MAX_K;
        for(int i = 0; i < MAX_K * MAX_K; i++)
        {
            partial.insert( (T)topk_tmp_val_buf[index + i], topk_tmp_id_buf[index + i]);
        }

        index = block_id * MAX_K;
        for(int i = 0; i < MAX_K; i++)
        {
            id_buf[index + i] = partial.p[i];
            val_buf[index + i] = partial.u[i];
        }
    }
}


template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE)
__global__ void batch_topk_kernel(
    const int * __restrict x,
    const T * __restrict y,
    int * __restrict z,
    T * __restrict v,
    int V,
    int K,
    T diversity_rate)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;

    // reposition x, y to data for the current vector
    x += vector_id * V;
    y += vector_id * V;

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    TopK<T, MAX_K> partial;
    for(int i = 0; i < MAX_K; ++i)
    {
        partial.p[i] = -1;
        partial.u[i] = -FLT_MAX;
    }
    for(int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE)
    {
        int i = elem_id % K;
        T elem = y[elem_id] + diversity_rate * (T) i;
        int elem_idx = elem_id; //x[elem_id];
        partial.insert(elem, elem_idx);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (thread_id == 0)
    {
        z += vector_id * K;
        v += vector_id * K;
        
        for(int i = 0; i < MAX_K; ++i)
        {
            if (i < K)
            {
                z[i] = x[total.p[i]];
                v[i] = y[total.p[i]];
            }
        }
    }
}

struct __align__(8) MD
{
    float m;
    float d;
};

__device__ __forceinline__ MD reduce_md_op(MD a, MD b)
{
    bool a_bigger = (a.m > b.m);
    MD bigger_m = a_bigger ? a : b;
    MD smaller_m = a_bigger ? b : a;
    MD res;
    res.d = bigger_m.d + smaller_m.d * __expf(smaller_m.m - bigger_m.m);
    res.m = bigger_m.m;
    return res;
}

template<typename T, int MAX_K>
struct TopKMD
{
    MD md;
    TopK<T, MAX_K> topk;
};

template<typename T, int MAX_K>
__device__ __forceinline__ TopKMD<T, MAX_K> reduce_topk_md_op(const TopKMD<T, MAX_K>& a, const TopKMD<T, MAX_K>& b)
{
    TopKMD<T, MAX_K> res;
    res.md = reduce_md_op(a.md, b.md);
    res.topk = reduce_topk_op(a.topk, b.topk);
    return res;
}

template<typename T, int ITEMS_PER_THREAD, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE)
__global__ void beam_online_softmax_topk_kernel(
    const T * __restrict x,
    const float * __restrict b,
    const T * __restrict c,
    const bool  * __restrict finished,
    int * __restrict z,
    T * __restrict v,
    int V,
    int K,
    int E)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;

    // reposition y to data for the current vector
    x += vector_id * V;

    typedef hipcub::BlockReduce<TopKMD<float, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    TopKMD<float, MAX_K> partial;
    bool finish = finished[vector_id];
    for(int i = 0; i < MAX_K; ++i)
    {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -FLT_MAX;
    }
    partial.md.m = -FLT_MAX;
    partial.md.d = 0.0F;

    if (finish)
    {
        for(int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE)
        {
            float elem = (elem_id == E) ? FLT_MAX : -FLT_MAX;
            MD new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
            //if (elem_id > THREADBLOCK_SIZE * MAX_K && (elem_id == E)) break;
        }
    }
    else
    {
        for(int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE)
        {
            float elem = x[elem_id] + b[elem_id];
            MD new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }

    TopKMD<float, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<float, MAX_K>);

    if (thread_id == 0)
    {
        z += vector_id * K;
        v += vector_id * K;
        c += vector_id;
        
        //float d_total_inverse = __fdividef(1.0F, total.md.d);
        float d_total_log = logf(total.md.d);
        for(int i = 0; i < MAX_K; ++i)
        {
            //float val = __expf(total.topk.u[i] - total.md.m) * d_total_inverse;
            float val = total.topk.u[i] - total.md.m - d_total_log;
            if (i < K)
            {
                z[i] = total.topk.p[i] + vector_id * V; // faster transformer needs absolute id
                v[i] = val + c[0];
            }
        }
    }
}

template<typename T, int ITEMS_PER_THREAD, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE)
__global__ void beam_online_softmax_topk_stage1_kernel(
    const T * __restrict x,
    const float * __restrict b,
    const bool  * __restrict finished,
    float * __restrict t,
    int V,
    int K,
    int E)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;

    const int PACKED_TOP_KMD_SIZE = 2 * MAX_K + 2;

    // one will have multiple sections per V
    const int v_local = (V + gridDim.y - 1) / gridDim.y;
    const int section_start = v_local * blockIdx.y;
    int section_end = section_start + v_local;
    section_end = (section_end > V)? V : section_end;

    // reposition x to data for the current vector
    x += vector_id * V;

    typedef hipcub::BlockReduce<TopKMD<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float buf_s[PACKED_TOP_KMD_SIZE]; // save intermediate result

    TopKMD<T, MAX_K> partial;
    bool finish = finished[vector_id];
    for(int i = 0; i < MAX_K; ++i)
    {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -FLT_MAX;
    }
    partial.md.m = -FLT_MAX;
    partial.md.d = 0.0F;

    if (finish)
    {
        for(int elem_id = section_start + thread_id; elem_id < section_end; elem_id += THREADBLOCK_SIZE)
        {
            float elem = (elem_id == E) ? FLT_MAX : -FLT_MAX;
            MD new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }
    else
    {
        for(int elem_id = section_start + thread_id; elem_id < section_end; elem_id += THREADBLOCK_SIZE)
        {
            T elem = x[elem_id] + b[elem_id];
            MD new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }

    TopKMD<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<T, MAX_K>);

    if (thread_id == 0)
    {
        for (int i = 0; i < K; i++)
        {
            reinterpret_cast<int *>(buf_s)[i] = total.topk.p[i] + vector_id * V; // faster transformer needs absolute id
            buf_s[MAX_K + i] = total.topk.u[i];
        }
        buf_s[2 * MAX_K] = total.md.d;
        buf_s[2 * MAX_K + 1] = total.md.m;
    }
    __syncthreads();
    if (threadIdx.x < PACKED_TOP_KMD_SIZE)
    {
        t[blockIdx.x * PACKED_TOP_KMD_SIZE * gridDim.y + blockIdx.y * PACKED_TOP_KMD_SIZE + threadIdx.x] = buf_s[threadIdx.x];
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE)
__global__ void beam_online_softmax_topk_stage2_kernel(
    const float * __restrict x,
    const T * __restrict c,
    int * __restrict z,
    T * __restrict v,
    int K,
    int parts_per_beam)
{
    const int vector_id = blockIdx.x;
    const int thread_id = threadIdx.x;
    const int PACKED_TOP_KMD_SIZE = 2 * MAX_K + 2;
    
    extern __shared__ char buf_s_[]; // intermediate result
    float * buf_s = reinterpret_cast<float *>(buf_s_);
    //__shared__ float buf_s[PACKED_TOP_KMD_SIZE * THREADBLOCK_SIZE]; // intermediate result

    typedef hipcub::BlockReduce<TopKMD<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    x += vector_id * PACKED_TOP_KMD_SIZE * parts_per_beam;

    TopKMD<T, MAX_K> partial;
    for(int i = 0; i < MAX_K; ++i)
    {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -FLT_MAX;
    }
    partial.md.m = -FLT_MAX;
    partial.md.d = 0.0F;

    // load and unpack into registers through smem
    for (int idx = thread_id; idx < PACKED_TOP_KMD_SIZE * parts_per_beam; idx += THREADBLOCK_SIZE)
    {
        buf_s[idx] = x[idx];
    }
    __syncthreads();

    if (threadIdx.x < parts_per_beam)
    {
        float * b_s = buf_s + thread_id * PACKED_TOP_KMD_SIZE;
        for (int i = 0; i < K; i++)
        {
            partial.topk.p[i] = reinterpret_cast<int *>(b_s)[i];
            partial.topk.u[i] = b_s[MAX_K + i];
        }
        partial.md.d = b_s[2 * MAX_K];
        partial.md.m = b_s[2 * MAX_K + 1];
    }
    __syncthreads();

    TopKMD<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<T, MAX_K>);

    if (thread_id == 0)
    {
        z += vector_id * K;
        v += vector_id * K;
        c += vector_id;
        
        float d_total_log = logf(total.md.d);
        for(int i = 0; i < MAX_K; ++i)
        {
            float val = total.topk.u[i] - total.md.m - d_total_log;
            if (i < K)
            {
                z[i] = total.topk.p[i];
                v[i] = val + c[0]; 
            }
        }
    }
}

template<typename T, int MAX_K>
void beam_online_softmax_topk_stage2_kernelLauncher(
    const float * temp_storage,
    const T * cum_log_probs,
    int * ids,
    T * vals,
    int batch_size,
    int beam_width,
    int parts_per_beam,
    hipStream_t stream)
{
    // might rewrite beam_online_softmax_topk_stage2_kernel no to depend on constant block size
    // in oreder to reduce compilation time
    int smem_stage2_size = parts_per_beam * (2 * MAX_K + 2) * sizeof(T);

    if (parts_per_beam <= 32)
    {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 32>
        <<<batch_size * beam_width, 32, smem_stage2_size, stream>>>
                (temp_storage, cum_log_probs, ids, vals,
                 beam_width, parts_per_beam);
        return;
    }
    if (parts_per_beam <= 64)
    {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 64>
        <<<batch_size * beam_width, 64, smem_stage2_size, stream>>>
                (temp_storage, cum_log_probs, ids, vals,
                 beam_width, parts_per_beam);
        return;
    }
    if (parts_per_beam <= 128)
    {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 128>
        <<<batch_size * beam_width, 128, smem_stage2_size, stream>>>
                (temp_storage, cum_log_probs, ids, vals,
                 beam_width, parts_per_beam);
        return;
    }
    assert(0);
}

template <typename T, int MAX_K>
void topK_softMax_kernelLauncher(const T* log_probs,
                                 const float* bias,
                                 const bool* finished,
                                 T* cum_log_probs,
                                 int* ids,
                                 void* temp_storage,
                                 const int temp_storage_size,
                                 const int batch_size, 
                                 const int beam_width, 
                                 const int vocab_size, 
                                 const int end_id,
                                 T diversity_rate,
                                 hipStream_t stream)
{
    const int items_per_thread = 1;
    const int block_sz = SMALL_TOP_K_SOFTMAX_THREADBLOCK_SIZE;

    assert(temp_storage_size % 2 == 0);
    assert(temp_storage_size >= 2 * batch_size * beam_width * beam_width);

    int* topk_tmp_id_buf = reinterpret_cast<int *>(temp_storage);
    T* topk_tmp_val_buf = reinterpret_cast<T *>(topk_tmp_id_buf + batch_size * beam_width * beam_width);
    float* tmp_buffer = reinterpret_cast<float *>(topk_tmp_val_buf + batch_size * beam_width * beam_width);

#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
    int voc_parts = 4;
    if (batch_size * beam_width < 256)
    {
        voc_parts = (256 + batch_size * beam_width - 1) / (batch_size * beam_width);
        voc_parts = std::min(128, voc_parts); // we implment up to 128
    }
    dim3 grid(batch_size * beam_width, voc_parts);
    beam_online_softmax_topk_stage1_kernel<T, items_per_thread, MAX_K, block_sz>
                            <<<grid, block_sz,0,stream>>>
                            (log_probs, bias, finished, tmp_buffer,
                            vocab_size, beam_width, end_id);
#endif
    if (beam_width > 1)
    {
#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
        beam_online_softmax_topk_stage2_kernelLauncher<T, MAX_K>
                                (tmp_buffer, cum_log_probs, topk_tmp_id_buf, topk_tmp_val_buf,
                                 batch_size, beam_width, voc_parts, stream);
#else
        beam_online_softmax_topk_kernel<T, items_per_thread, MAX_K, block_sz>
                        <<<batch_size * beam_width, block_sz, 0, stream>>>
                                (log_probs, bias, cum_log_probs, finished, topk_tmp_id_buf, 
                                topk_tmp_val_buf, vocab_size, beam_width, end_id);
#endif
#if 0
         // wrong result with diversity_rate != 0.f
         batch_topK_kernel<T, MAX_K, 32><<<batch_size, 32, 0, stream>>>
                                (topk_tmp_id_buf, topk_tmp_val_buf, ids, cum_log_probs);
#else
         batch_topk_kernel<T, MAX_K, 32><<<batch_size, 32, 0, stream>>>
                                (topk_tmp_id_buf, topk_tmp_val_buf,
                                ids, cum_log_probs, beam_width * beam_width, beam_width, diversity_rate);
#endif
    }
    else
    {
#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
        beam_online_softmax_topk_stage2_kernelLauncher<T, MAX_K>
                                (tmp_buffer, cum_log_probs, ids, cum_log_probs,
                                batch_size, beam_width, voc_parts, stream);
#else
        beam_online_softmax_topk_kernel<T, items_per_thread, MAX_K, block_sz>
                            <<<batch_size * beam_width, block_sz, 0, stream>>>
                                   (log_probs, bias, cum_log_probs, finished, ids, 
                                    cum_log_probs, vocab_size, beam_width, end_id);
#endif
    }
}

#define CASE_K(K) \
  case K : \
    topK_softMax_kernelLauncher<T, K> \
      (log_probs, bias, finished, cum_log_probs, ids, temp_storage, temp_storage_size, \
      batch_size, beam_width, vocab_size, end_id, diversity_rate, stream); \
  break; \

template <typename T>
void topK_softMax(const T* log_probs, 
                  const float* bias, 
                  const bool* finished, 
                  T* cum_log_probs,
                  int* ids,
                  void* temp_storage,
                  DecodingBeamsearchArguments args,
                  hipStream_t stream)
{
    const int temp_storage_size = args.temp_storage_size_;
    const int batch_size = args.batch_size_;
    const int beam_width = args.beam_width_;
    const int vocab_size = args.vocab_size_;
    const int end_id = args.end_id_;
    const T diversity_rate = args.beam_search_diversity_rate_;

    switch(beam_width)
    {
        CASE_K(1);
        CASE_K(2);
        CASE_K(4);
        default :
            printf("[ERROR] Topk kernel does not support beamwidth = %d \n", beam_width);
            exit(0);
            break;
    }
}
#undef CASE_K

template void topK_softMax<float>(const float* log_probs, 
                                const float* bias, 
                                const bool* finished, 
                                float* cum_log_probs,
                                int* ids, 
                                void * tmp_storage,
                                DecodingBeamsearchArguments args,
                                hipStream_t stream);
} // end of namespace fastertransformer