#include "hip/hip_runtime.h"
/*
* Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
/**
* Open sourced multi-head attention
**/

#include "fastertransformer/allocator.h"
#include "fastertransformer/cuda/multi_head_attention.h"
#include "fastertransformer/cuda/open_attention.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
namespace fastertransformer{
namespace cuda{

/**
* Multi-head attetion open sourced
*/
#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  #pragma unroll
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);
                              
  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  #pragma unroll
  for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : -1e20f;
  val = warpReduceMax(val);

  return val;
}

  __inline__ __device__
int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
  return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

//build a mapping for fullData to removePaddingData
//grid((valid_word_num+63)/64)
//block(64)
__global__ void mappingRemovePaddingData(int *mapping, const int* sequence_id_offset, const int valid_word_num){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < valid_word_num)
    mapping[idx + __ldg(sequence_id_offset + idx)] = idx;
}

//add_QK_bias_transform for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = batch_size * seq_len, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C
//only for int32 input & int8 output
//seq_len, size_per_head must be a multiple of 32
//grid.x = batch_size * seq_len * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform(int8_t *q_buf_, int8_t *k_buf_, const int32_t* Q, const T* bias_Q, 
                           const int32_t* K, const T* bias_K, const int m, const int batch_size, 
                           const int seq_len, const int head_num, const int size_per_head, int stride, 
                           const float * q_weight_amax, const float *q_input_deQFactor_div127_ptr, const float * k_weight_amax, 
                           const float *k_input_deQFactor_div127_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                           bool use_ORDER_COL32_2R_4R4)
{
  const int32_t* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  const float* weight_amax;
  int qk_id = blockIdx.x / m;

  data_ptr = qk_id == 0 ? Q : K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  const float input_deQFactor_div127 = qk_id == 0 ? __ldg(q_input_deQFactor_div127_ptr) : __ldg(k_input_deQFactor_div127_ptr);
  weight_amax = qk_id == 0 ? q_weight_amax : k_weight_amax;
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int threadIdx4 = threadIdx.x << 2;
  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = blockIdx.x % seq_len;

  int data_id = (((threadIdx4 >> 5) << 5)*m + ((blockIdx.x%m) << 5) + (threadIdx4&31));

  float scale;
  float tmp;
  char4 tmp4;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4)* input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);


  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major

  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row;  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
               //COL32_2R_4R4
               (
               ((row_id >> 5) << 10) +
               //(((row%8)/2*4+row/8)*2+row%2)*32+col
               (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
               )
               ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

//add_QK_bias_transform for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = batch_size * seq_len, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C
//only for int8 IO
//seq_len, size_per_head must be a multiple of 32
//grid.x = batch_size * seq_len * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q, 
                           const int8_t* K, const T* bias_K, const int m, const int batch_size, 
                           const int seq_len, const int head_num, const int size_per_head, int stride, 
                           const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                           bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / m;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  const float input_deQFactor = qk_id == 0 ? __ldg(q_input_deQFactor_ptr) : __ldg(k_input_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int threadIdx4 = threadIdx.x << 2;
  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = blockIdx.x % seq_len;

  int data_id = (((threadIdx4 >> 5) << 5)*m + ((blockIdx.x%m) << 5) + (threadIdx4&31)) >> 2;

  float scale;
  float tmp;
  char4 tmp4 = __ldg(data_ptr+data_id);
  scale = static_cast<float>(tmp4.x) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * input_deQFactor;;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);


  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major

  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row;  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
               //COL32_2R_4R4
               (
               ((row_id >> 5) << 10) +
               //(((row%8)/2*4+row/8)*2+row%2)*32+col
               (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
               )
               ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

//add_QK_bias_transform & rebuild padding for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = valid_word_num, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32 input & int8 output
//seq_len, size_per_head must be a multiple of 32
//grid.x = valid_word_num * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_rebuild_padding(int8_t *q_buf_, int8_t *k_buf_, const int32_t* Q, const T* bias_Q, 
                                           const int32_t* K, const T* bias_K, const int* sequence_id_offset, 
                                           const int valid_word_num, const int m, const int batch_size, const int seq_len, 
                                           const int head_num, const int size_per_head, int stride, const float * q_weight_amax, 
                                           const float *q_input_deQFactor_div127_ptr, const float * k_weight_amax, 
                                           const float *k_input_deQFactor_div127_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                           bool use_ORDER_COL32_2R_4R4)
{
  const int32_t* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  const float* weight_amax;
  int qk_id = blockIdx.x / valid_word_num;

  data_ptr = qk_id == 0 ? Q : K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  
  int threadIdx4 = threadIdx.x << 2;
  int m_full_idx = blockIdx.x % valid_word_num;
  m_full_idx = (valid_word_num != m) ? (m_full_idx + __ldg(sequence_id_offset+m_full_idx)) : m_full_idx;
  int batch_id = m_full_idx / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = m_full_idx % seq_len;
  
  const float input_deQFactor_div127 = qk_id == 0 ? __ldg(q_input_deQFactor_div127_ptr) : __ldg(k_input_deQFactor_div127_ptr);
  weight_amax = qk_id == 0 ? q_weight_amax : k_weight_amax;
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int data_id = (((threadIdx4 >> 5) << 5)*valid_word_num + ((blockIdx.x%valid_word_num) << 5) + (threadIdx4&31));
    
  float scale;
  float tmp;
  char4 tmp4;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4)* input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);

  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major
  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row; 
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL32_2R_4R4
              (
              ((row_id >> 5) << 10) +
              //(((row%8)/2*4+row/8)*2+row%2)*32+col
              (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
              )
              ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

//add_QK_bias_transform & rebuild padding for batch int8 hipblasLtMatmul & per tensor quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = valid_word_num, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8 IO
//seq_len, size_per_head must be a multiple of 32
//grid.x = valid_word_num * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_rebuild_padding(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q, 
                                           const int8_t* K, const T* bias_K, const int* sequence_id_offset, 
                                           const int valid_word_num, const int m, const int batch_size, const int seq_len, 
                                           const int head_num, const int size_per_head, int stride,  
                                           const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr, 
                                           const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                           bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / valid_word_num;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  
  int threadIdx4 = threadIdx.x << 2;
  int m_full_idx = blockIdx.x % valid_word_num;
  m_full_idx = (valid_word_num != m) ? (m_full_idx + __ldg(sequence_id_offset+m_full_idx)) : m_full_idx;
  int batch_id = m_full_idx / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = m_full_idx % seq_len;
  
  const float deQFactor = qk_id == 0 ? __ldg(q_deQFactor_ptr) : __ldg(k_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int data_id = (((threadIdx4 >> 5) << 5)*valid_word_num + ((blockIdx.x%valid_word_num) << 5) + (threadIdx4&31)) >> 2;
    
  float scale;
  float tmp;
  char4 tmp4;
  
  tmp4 = __ldg(data_ptr+data_id);
  
  scale = static_cast<float>(tmp4.x) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);

  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major
  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row; 
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL32_2R_4R4
              (
              ((row_id >> 5) << 10) +
              //(((row%8)/2*4+row/8)*2+row%2)*32+col
              (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
              )
              ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

//input matrix a matrix of m = batch_size*seq_len , n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32_t Input int8_t Output
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per axis quantization for weight
template <typename T>
__global__
void add_V_bias_transform(int8_t *v_buf_, const int32_t *V, const T *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, int stride, const float* weight_amax, 
                          const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col = head_id*size_per_head + id_in_size;
  int row = batch_id*seq_len + word_id;
  int inIdx = (((col >> 5) << 5)*batch_size*seq_len + ((row << 5) + (col&31)));
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  float tmp;
  float scale;

  //const half2* bias_ptr2 = (const half2*)bias_ptr;
  //half2 tmp2;

  //tmp2 = __ldg(&bias_ptr2[col >> 1]);
  
  scale = __ldg(data_ptr + inIdx) * __ldg(weight_amax + col) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr + col));//(tmp2.x);
  shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
  scale = __ldg(data_ptr + inIdx + 1) * __ldg(weight_amax + col + 1) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));//(tmp2.y);
  shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);
  
  //tmp2 = __ldg(&bias_ptr2[(col >> 1) + 1]);

  scale = __ldg(data_ptr+inIdx+2) * __ldg(weight_amax+col+2) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));//(tmp2.x);
  shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
  scale = __ldg(data_ptr+inIdx + 3) * __ldg(weight_amax+col+3) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));//(tmp2.y);
  shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);

  __syncthreads();

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          ((((id_in_size >> 3) << 3) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }

        
  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}



//input matrix a matrix of m = batch_size*seq_len , n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8_t IO
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per tensor quantization for weight
template <typename T>
__global__
void add_V_bias_transform(int8_t *v_buf_, const int8_t *V, const T *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, int stride,
                          const float *input_deQFactor_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor = __ldg(input_deQFactor_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const char4* data_ptr = (const char4*)V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col = head_id*size_per_head + id_in_size;
  int row = batch_id*seq_len + word_id;
  int inIdx = (((col >> 5) << 5)*batch_size*seq_len + ((row << 5) + (col&31))) >> 2;
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  float tmp;
  float scale;

  //const half2* bias_ptr2 = (const half2*)bias_ptr;
  //half2 tmp2;

  //tmp2 = __ldg(&bias_ptr2[col >> 1]);
  
  char4 dataTmp = __ldg(data_ptr + inIdx);
  
  scale = dataTmp.x * input_deQFactor;
  tmp = scale + static_cast<float>(__ldg(bias_ptr + col));//(tmp2.x);
  shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
  scale = dataTmp.y * input_deQFactor;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));//(tmp2.y);
  shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);
  
  //tmp2 = __ldg(&bias_ptr2[(col >> 1) + 1]);

  scale = dataTmp.z * input_deQFactor;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));//(tmp2.x);
  shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
  scale = dataTmp.w * input_deQFactor;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));//(tmp2.y);
  shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);

  __syncthreads();

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          ((((id_in_size >> 3) << 3) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }

  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <>
__global__
void add_V_bias_transform(int8_t *v_buf_, const int32_t *V, const half *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, int stride, const float* weight_amax, 
                          const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  
  int blockIdy32 = (blockIdx.y << 5);
  int blockIdx32 = (blockIdx.x << 5);
  int word_id = blockIdy32 + threadIdx.y;
  int id_in_size = blockIdx32 + threadIdx4;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col = head_id*size_per_head + id_in_size;
  int row = batch_id*seq_len + word_id;
  int inIdx = ((col & 0xffffffe0)*batch_size*seq_len + ((row << 5) + (col&31)));
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  int col_2 = col >> 1;
  float scale;

  const half2* bias_ptr2 = (const half2*)V_bias;
  half2 tmp2;

  tmp2 = __ldg(bias_ptr2+col_2);
  
  scale = __ldg(data_ptr+inIdx) * __ldg(weight_amax+col) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.x);
  shm[sh_row][sh_col] = float_to_int8_rn(scale*out_scale);
  
  scale = __ldg(data_ptr+inIdx+1) * __ldg(weight_amax+col+1) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.y);
  shm[sh_row][sh_col+1] = float_to_int8_rn(scale*out_scale);
  
  tmp2 = __ldg(bias_ptr2 + col_2 + 1);

  scale = __ldg(data_ptr + inIdx + 2) * __ldg(weight_amax + col + 2) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.x);
  shm[sh_row][sh_col+2] = float_to_int8_rn(scale*out_scale);
  
  scale = __ldg(data_ptr + inIdx + 3) * __ldg(weight_amax + col + 3) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.y);
  shm[sh_row][sh_col+3] = float_to_int8_rn(scale*out_scale);

  __syncthreads();

  //for dst of (size_per_head, seq_len)
  word_id = blockIdy32 + threadIdx4;
  id_in_size = blockIdx32 + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

//add bias into V & rebuild padding 
//input matrix a matrix of m = valid_word_num, n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32_t Input int8_t Output
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per axis quantization for weight
template <typename T>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int32_t *V, const T *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float* weight_amax, const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row != -1){
    col = head_id*size_per_head + id_in_size;  
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31)));
  
    float tmp;
    float scale;
  
    const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
    const float out_scale = __ldg(out_scale_ptr);
  
    scale = __ldg(data_ptr + inIdx) * __ldg(weight_amax + col) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
    scale = __ldg(data_ptr + inIdx + 1) * __ldg(weight_amax + col + 1) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = __ldg(data_ptr+inIdx+2) * __ldg(weight_amax+col+2) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
    scale = __ldg(data_ptr+inIdx + 3) * __ldg(weight_amax+col+3) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int32_t *V, const half *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float* weight_amax, const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  
  int blockIdy32 = (blockIdx.y << 5);
  int blockIdx32 = (blockIdx.x << 5);
  int word_id = blockIdy32 + threadIdx.y;
  int id_in_size = blockIdx32 + threadIdx4;

  
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row >= 0){
    const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
    const float out_scale = __ldg(out_scale_ptr);
    col = head_id*size_per_head + id_in_size;
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31)));
    int col_2 = col >> 1;
    float scale;

    const half2* bias_ptr2 = (const half2*)V_bias;
    half2 tmp2;

    tmp2 = __ldg(bias_ptr2+col_2);
  
    scale = __ldg(data_ptr+inIdx) * __ldg(weight_amax+col) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.x);
    shm[sh_row][sh_col] = float_to_int8_rn(scale*out_scale);
  
    scale = __ldg(data_ptr+inIdx+1) * __ldg(weight_amax+col+1) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.y);
    shm[sh_row][sh_col+1] = float_to_int8_rn(scale*out_scale);
  
    tmp2 = __ldg(bias_ptr2 + col_2 + 1);

    scale = __ldg(data_ptr + inIdx + 2) * __ldg(weight_amax + col + 2) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.x);
    shm[sh_row][sh_col+2] = float_to_int8_rn(scale*out_scale);
  
    scale = __ldg(data_ptr + inIdx + 3) * __ldg(weight_amax + col + 3) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.y);
    shm[sh_row][sh_col+3] = float_to_int8_rn(scale*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = blockIdy32 + threadIdx4;
  id_in_size = blockIdx32 + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

//add bias into V & rebuild padding 
//input matrix a matrix of m = valid_word_num, n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8_t IO
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per tensor quantization for weight
template <typename T>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int8_t *V, const T *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float *deQFactor_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const char4* data_ptr = (const char4*)V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row != -1){
    col = head_id*size_per_head + id_in_size;  
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31))) >> 2;
  
    float tmp;
    float scale;
  
    const float deQFactor = __ldg(deQFactor_ptr);
    const float out_scale = __ldg(out_scale_ptr);
  
    char4 dataTmp = __ldg(data_ptr + inIdx);
  
    scale = dataTmp.x * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.y * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.z * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.w * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;  
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

__global__
void trt_add_QKV_bias(half2* Q, const half2* bias_Q, half2* K, const half2* bias_K, half2* V, const half2* bias_V, 
  half2* q_buf_, half2* k_buf_, half2* v_buf_, 
  const int valid_word_num, const int head_num, const int size_per_head)
{
  // Add bias, and then transpose from 
  // [3, valid_word_num, head, size] -> [valid_word_num, head, 3, size]
  
  // const int seq_id = blockIdx.x % valid_word_num;
  // const int qkv_id = (blockIdx.x - seq_id) / valid_word_num;
  const int seq_id = blockIdx.x;
  const int size_id = threadIdx.x % size_per_head;
  const int head_id = (threadIdx.x - size_id) / size_per_head;

  const int target_offset = blockIdx.x * head_num * 3 * size_per_head + head_id * 3 * size_per_head;

  q_buf_[ target_offset + 
          0 * size_per_head +
          size_id] = Q[ seq_id * blockDim.x + threadIdx.x] + bias_Q[threadIdx.x];

  q_buf_[ target_offset + 
          1 * size_per_head +
          size_id] = K[ seq_id * blockDim.x + threadIdx.x] + bias_K[threadIdx.x];

  q_buf_[ target_offset + 
          2 * size_per_head +
          size_id] = V[ seq_id * blockDim.x + threadIdx.x] + bias_V[threadIdx.x];
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::trt_add_QKV_bias_kernelLauncher(
  const DataType_* bias_Q,
  const DataType_* bias_K,
  const DataType_* bias_V)
{
  dim3 grid;
  dim3 block;

  grid.x = param_.valid_word_num;
  block.x = head_num_ * size_per_head_ / 2;

  assert(block.x <= 1024);

  trt_add_QKV_bias<<<grid, block, 0, param_.stream>>>((half2*)query_buf_, (const half2*)bias_Q, 
                                                      (half2*)key_buf_, (const half2*)bias_K, 
                                                      (half2*)value_buf_, (const half2*)bias_V, 
                                                      (half2*)q_buf_, (half2*)k_buf_, (half2*)v_buf_,
                                                      param_.valid_word_num, 
                                                      head_num_, size_per_head_ / 2);
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::fused_multiHeadAttr_kernelLauncher()
{
  trt_add_QKV_bias_kernelLauncher(param_.self_attention.query_weight.bias,
                                  param_.self_attention.key_weight.bias,
                                  param_.self_attention.value_weight.bias);


  const int B = param_.trt_seqlen_size - 1;
  const int maxS = from_seq_len_;
  int S = 384;
  if (maxS <= 64)
  {
    S = 64;
  }
  else if (maxS <= 96)
  {
    S = 96;
  }
    else if (maxS <= 128)
  {
    S = 128;
  }
  else if (maxS <= 256)
  {
    S = 256;
  }
  dispatcher_fp16->setup(S, B);
  dispatcher_fp16->run(q_buf_, nullptr, param_.trt_seqlen_offset, trt_attn_workspace_, param_.attr_out, param_.stream);
}

template<typename T>
__global__
void add_QKV_bias(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{

  T* data_ptr;
  T* buf_ptr;
  const T* bias_ptr;
  
  int m = batch_size * seq_len;
  int n = head_num * size_per_head;

  int qkv_id = blockIdx.x * word_per_block / m;
  int row_offset = (blockIdx.x * word_per_block % m) * n;

  if(qkv_id == 0)
  {
    data_ptr = Q + row_offset;
    buf_ptr = q_buf_;
    bias_ptr = bias_Q;
  }
  else if(qkv_id == 1)
  {
    data_ptr = K + row_offset;
    buf_ptr = k_buf_;
    bias_ptr = bias_K;
  }
  else
  {
    data_ptr = V + row_offset;
    buf_ptr = v_buf_;
    bias_ptr = bias_V;
  }

  int batch_id = (blockIdx.x * word_per_block % m) / seq_len;
  int head_id = threadIdx.x / size_per_head;
  int id_in_head = threadIdx.x % size_per_head;
  int word_start_id = (blockIdx.x * word_per_block) % seq_len;

  T bias = __ldg(&bias_ptr[threadIdx.x]);

  for(int i = word_start_id; i < word_start_id + word_per_block; ++i)
  {
    T tmp = data_ptr[threadIdx.x] + bias;

    int target_id = batch_id * (seq_len * head_num * size_per_head) + head_id * seq_len * size_per_head + 
      i * size_per_head + id_in_head;

    buf_ptr[target_id] = tmp;
    data_ptr += n;
  }
}

template <>
__global__
void add_QKV_bias(half* Q, const half* bias_Q, half* K, const half* bias_K, half* V, const half* bias_V, 
  half* q_buf_, half* k_buf_, half* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_id = tid / (head_num * seq_len * size_per_head);
  int seq_id = (tid % (head_num * seq_len * size_per_head)) / (head_num * size_per_head);
  int head_id = (tid % (head_num * size_per_head)) / size_per_head;
  int id = tid % size_per_head;
  int target_id = target_index(batch_id, seq_id, head_id, id, batch_size, seq_len, head_num, size_per_head);

  int bias_id = threadIdx.x;

  half2* src_ptr = (half2*)Q;
  half2* dst_ptr = (half2*)q_buf_;
  const half2* bias_ptr = (const half2*)bias_Q;

  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)K;
  dst_ptr = (half2*)k_buf_;
  bias_ptr = (const half2*)bias_K;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)V;
  dst_ptr = (half2*)v_buf_;
  bias_ptr = (const half2*)bias_V;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));
}

template<typename T>
__global__
void add_QKV_bias_rebuild_padding(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int* mask_offset)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int bdim = blockDim.x;

  const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
  const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
  const int tgt_head_id = tid / size_per_head;
  const int tgt_hidden_id = tid % size_per_head;

  const int src_id = bid * bdim + tid;
  const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + \
                    tgt_head_id * seq_len * size_per_head + \
                    tgt_seq_id * size_per_head + \
                    tgt_hidden_id;
  
  q_buf_[tgt_id] = Q[src_id] + bias_Q[tid];
  k_buf_[tgt_id] = K[src_id] + bias_K[tid];
  v_buf_[tgt_id] = V[src_id] + bias_V[tid];
}

template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, 
  const T scalar)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;
    int mask_offset = batch_id * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
      mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val): -1e20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
      mask_offset += seq_len;
    }
}


template <typename T>
__global__
void softmax_kernel_v2(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, 
  const int seq_len, const float scalar)
{
    int batch_id = blockIdx.x / head_num / seq_len;
    int seq_id = blockIdx.x % seq_len;
    int qk_offset = blockIdx.x * seq_len;
    int mask_offset = batch_id * seq_len * seq_len + seq_id * seq_len;

    __shared__ float s_sum, s_max;

    float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
    mask_val = (1.0f - mask_val) * -10000.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);
    if(threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);

    if(threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if(threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len + 31)/32*32)
template <typename T>
__global__
void softmax_kernel_v3(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
    
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    float tmp = -1e20f;
    int qk_offset;
    __shared__ float s_mean, s_max;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = qk * static_cast<float>(scalar) + mask_val;
    }

    float max_val = blockReduceMax<float>(tmp);
    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    float qk_tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);
    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();
    
    if(qual)
      qk_buf_[qk_offset] = (T)(qk_tmp * s_mean);
  }
}  


//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//seq_len % 2 == 0
template <>
__global__
void softmax_kernel_v3(half* qk_buf_, const half* attr_mask, 
                      const int batch_size, const int head_num, 
                      const int seq_len, const half scalar)
{
  int threadIdx2 = threadIdx.x << 1;
  bool qual = threadIdx2 < seq_len;
  half2* qk_buf_half2Ptr = (half2*) qk_buf_;
  const half2* attr_mask_half2Ptr = (const half2*) attr_mask;
  __shared__ float s_mean, s_max;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    half2 tmp = __float2half2_rn(0.0f);

    float max_val = -1e20f;
    half2 qk;
    if (qual){ 
      qk_offset = ((((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len) >> 1) + threadIdx.x;
      int mask_offset = (((blockIdx.y * seq_len + seq_id) * seq_len) >> 1) + threadIdx.x;

      qk = qk_buf_half2Ptr[qk_offset];
      half2 mask_val = __ldg(&attr_mask_half2Ptr[mask_offset]);
      half2 mask_val_tmp = __hmul2(__hsub2(__float2half2_rn(1.0f), mask_val), __float2half2_rn(-10000.0f));
      tmp = __hadd2(__hmul2(__half2half2(scalar), qk), mask_val_tmp);
      max_val = fmax((float)tmp.x, (float)tmp.y);
    }
    
    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    if (qual){
      tmp = h2exp(__hsub2(tmp, __float2half2_rn(s_max)));
    }
    float sum_val = blockDim.x <= 32 ? warpReduceSum((float)(tmp.x + tmp.y)) : blockReduceSum<float>((float)(tmp.x + tmp.y));

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual){
      qk = __hmul2(tmp, __float2half2_rn(s_mean));
      qk_buf_half2Ptr[qk_offset] = qk;
    }
  }
}

//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len + 31)/32*32)
//for seq_len not larger than 32
template <typename T>
__global__
void softmax_kernel_v3_LE32(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    __shared__ float s_mean, s_max;
    float tmp = -1e20f;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = static_cast<float>(qk) * static_cast<float>(scalar) + mask_val;
    }
    float max_val = warpReduceMax<float>(tmp);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = warpReduceSum<float>(tmp);

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual)
      qk_buf_[qk_offset] = (T)(tmp * s_mean);
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/4 + 31)/32*32)
//for int32_t I; int8 O;
template <typename T>
__global__
void softmax_COL32(int8_t* qk_buf_, const int32_t* int_buf, const T* attr_mask, const int batch_size, 
                   const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                   const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdx4 = threadIdx.x << 2;

  char4* buf4Ptr = (char4 *)qk_buf_;

  bool qual = threadIdx4 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char4 tmp4 = {0, 0, 0, 0};
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx4 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx4 & 31);
                
    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf4Ptr[inIdx >> 2] = tmp4;
      continue;
    }  

    float4 floatTmp4 = {0.0f, 0.0f, 0.0f, 0.0f};    

    if (qual){
      floatTmp4.x = static_cast<float>(__ldg(int_buf + inIdx)) * scalar1;
      floatTmp4.y = static_cast<float>(__ldg(int_buf+inIdx+1)) * scalar1;
      floatTmp4.z = static_cast<float>(__ldg(int_buf+inIdx+2)) * scalar1;
      floatTmp4.w = static_cast<float>(__ldg(int_buf+inIdx+3)) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx4 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp4.x = floatTmp4.x + mask_val;
      max_val = fmaxf(max_val, floatTmp4.x);

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp4.y = floatTmp4.y + mask_val;
      max_val = fmaxf(max_val, floatTmp4.y);

      //for z
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+2))) * -10000.0f;
      floatTmp4.z = floatTmp4.z + mask_val;
      max_val = fmaxf(max_val, floatTmp4.z);

      //for w
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+3))) * -10000.0f;
      floatTmp4.w = floatTmp4.w + mask_val;
      max_val = fmaxf(max_val, floatTmp4.w);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp4.x = __expf(floatTmp4.x - s_max);
      sum_val += floatTmp4.x;
      floatTmp4.y = __expf(floatTmp4.y - s_max);
      sum_val += floatTmp4.y;
      floatTmp4.z = __expf(floatTmp4.z - s_max);
      sum_val += floatTmp4.z;
      floatTmp4.w = __expf(floatTmp4.w - s_max);
      sum_val += floatTmp4.w;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){

      tmp4.x = float_to_int8_rn(floatTmp4.x*s_sum);
      tmp4.y = float_to_int8_rn(floatTmp4.y*s_sum);
      tmp4.z = float_to_int8_rn(floatTmp4.z*s_sum);
      tmp4.w = float_to_int8_rn(floatTmp4.w*s_sum);

      buf4Ptr[inIdx >> 2] = tmp4;
    }
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/4 + 31)/32*32)
//for int8_t IO;
template <typename T>
__global__
void softmax_COL32(int8_t* qk_buf_, const int8_t* int_buf, const T* attr_mask, const int batch_size, 
                   const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                   const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdx4 = threadIdx.x << 2;

  char4* buf4Ptr = (char4 *)qk_buf_;
  const char4* inBuf4Ptr = (const char4*)int_buf;

  bool qual = threadIdx4 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
  
    char4 tmp4 = {0, 0, 0, 0};
    int inIdx = ((blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx4 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx4 & 31)) >> 2;
  
    //set softmax of padding word to 0
    const float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf4Ptr[inIdx] = tmp4;
      continue;
    }
    
    float4 floatTmp4 = {0.0f, 0.0f, 0.0f, 0.0f};

    if (qual){
      tmp4 = __ldg(inBuf4Ptr + inIdx);
      floatTmp4.x = static_cast<float>(tmp4.x) * scalar1;
      floatTmp4.y = static_cast<float>(tmp4.y) * scalar1;
      floatTmp4.z = static_cast<float>(tmp4.z) * scalar1;
      floatTmp4.w = static_cast<float>(tmp4.w) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx4 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp4.x = floatTmp4.x + mask_val;
      max_val = fmaxf(max_val, floatTmp4.x);

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp4.y = floatTmp4.y + mask_val;
      max_val = fmaxf(max_val, floatTmp4.y);

      //for z
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+2))) * -10000.0f;
      floatTmp4.z = floatTmp4.z + mask_val;
      max_val = fmaxf(max_val, floatTmp4.z);

      //for w
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+3))) * -10000.0f;
      floatTmp4.w = floatTmp4.w + mask_val;
      max_val = fmaxf(max_val, floatTmp4.w);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp4.x = __expf(floatTmp4.x - s_max);
      sum_val += floatTmp4.x;
      floatTmp4.y = __expf(floatTmp4.y - s_max);
      sum_val += floatTmp4.y;
      floatTmp4.z = __expf(floatTmp4.z - s_max);
      sum_val += floatTmp4.z;
      floatTmp4.w = __expf(floatTmp4.w - s_max);
      sum_val += floatTmp4.w;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){

      tmp4.x = float_to_int8_rn(floatTmp4.x*s_sum);
      tmp4.y = float_to_int8_rn(floatTmp4.y*s_sum);
      tmp4.z = float_to_int8_rn(floatTmp4.z*s_sum);
      tmp4.w = float_to_int8_rn(floatTmp4.w*s_sum);

      buf4Ptr[inIdx] = tmp4;
    }
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = (seq_len + 31)/32
//for int32_t I; int8 O;
//for seq_len <= 32
template <typename T>
__global__
void softmax_COL32_LE32(int8_t* qk_buf_, const int32_t* int_buf, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdxx = threadIdx.x;
  bool qual = threadIdxx < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdxx & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdxx & 31);
  
    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        qk_buf_[inIdx] = 0;
      continue;
    }

    float floatTmp = qual ? static_cast<float>(__ldg(int_buf + inIdx)) * scalar1 : 0.0f;

    float mask_val, max_val;

    __shared__ float s_max, s_sum;

    mask_id = qual ? threadIdxx + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len : 0;
    mask_val = qual ? (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f : 0.0f;
    floatTmp = qual ? floatTmp + mask_val : 0.0f;
    max_val = qual ? floatTmp : -1e20f;

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    floatTmp = qual ? __expf(floatTmp - s_max) : 0.0f;
    
    float sum_val = blockDim.x <= 32 ? warpReduceSum(floatTmp) : blockReduceSum<float>(floatTmp);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    
    if (qual){
      qk_buf_[inIdx] = float_to_int8_rn(floatTmp*s_sum);
    }
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = (seq_len + 31)/32
//for int8_t IO;
//for seq_len <= 32
template <typename T>
__global__
void softmax_COL32_LE32(int8_t* qk_buf_, const int8_t* int_buf, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdxx = threadIdx.x;
  bool qual = threadIdxx < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdxx & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdxx & 31);

    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        qk_buf_[inIdx] = 0;
      continue;
    }


    float floatTmp = qual ? static_cast<float>(__ldg(int_buf + inIdx)) * scalar1 : 0.0f;

    float mask_val, max_val;

    __shared__ float s_max, s_sum;

    mask_id = qual ? threadIdxx + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len : 0;
    mask_val = qual ? (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f : 0.0f;
    floatTmp = qual ? floatTmp + mask_val : 0.0f;
    max_val = qual ? floatTmp : -1e20f;

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    floatTmp = qual ? __expf(floatTmp - s_max) : 0.0f;
    
    float sum_val = blockDim.x <= 32 ? warpReduceSum(floatTmp) : blockReduceSum<float>(floatTmp);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    
    if (qual){
      qk_buf_[inIdx] = float_to_int8_rn(floatTmp*s_sum);
    }
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//for int32_t I; int8 O;
//for seq_len in (32, 64]
template <typename T>
__global__
void softmax_COL32_LE64(int8_t* qk_buf_, const int32_t* int_buf, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdx2 = threadIdx.x << 1;

  char2* buf2Ptr = (char2 *)qk_buf_;

  bool qual = threadIdx2 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char2 tmp2 = {0, 0};
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx2 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx2 & 31);

    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf2Ptr[inIdx >> 1] = tmp2;
      continue;
    }

    float2 floatTmp2 = {0.0f, 0.0f};
    if (qual){
      floatTmp2.x = static_cast<float>(__ldg(int_buf + inIdx)) * scalar1;
      floatTmp2.y = static_cast<float>(__ldg(int_buf + inIdx + 1)) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx2 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp2.x = floatTmp2.x + mask_val;

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp2.y = floatTmp2.y + mask_val;
            
      max_val = fmaxf(floatTmp2.x, floatTmp2.y);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp2.x = __expf(floatTmp2.x - s_max);
      sum_val += floatTmp2.x;
      floatTmp2.y = __expf(floatTmp2.y - s_max);
      sum_val += floatTmp2.y;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){
      tmp2.x = float_to_int8_rn(floatTmp2.x*s_sum);
      tmp2.y = float_to_int8_rn(floatTmp2.y*s_sum);
      buf2Ptr[inIdx >> 1] = tmp2;
    }
  }
}

//int_buf are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//for int8_t IO
//for seq_len in (32, 64]
template <typename T>
__global__
void softmax_COL32_LE64(int8_t* qk_buf_, const int8_t* int_buf, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdx2 = threadIdx.x << 1;

  char2* buf2Ptr = (char2 *)qk_buf_;
  const char2* inBuf2Ptr = (const char2 *)int_buf;

  bool qual = threadIdx2 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char2 tmp2 = {0, 0};
    int inIdx = ((blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx2 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx2 & 31)) >> 1;

    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf2Ptr[inIdx] = tmp2;
      continue;
    }

    float2 floatTmp2 = {0.0f, 0.0f};
    if (qual){
      tmp2 = __ldg(inBuf2Ptr + inIdx);
      floatTmp2.x = static_cast<float>(tmp2.x) * scalar1;
      floatTmp2.y = static_cast<float>(tmp2.y) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx2 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp2.x = floatTmp2.x + mask_val;

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp2.y = floatTmp2.y + mask_val;
            
      max_val = fmaxf(floatTmp2.x, floatTmp2.y);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp2.x = __expf(floatTmp2.x - s_max);
      sum_val += floatTmp2.x;
      floatTmp2.y = __expf(floatTmp2.y - s_max);
      sum_val += floatTmp2.y;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){
      tmp2.x = float_to_int8_rn(floatTmp2.x*s_sum);
      tmp2.y = float_to_int8_rn(floatTmp2.y*s_sum);
      buf2Ptr[inIdx] = tmp2;
    }
  }
}



template<typename T>
__global__
void transpose(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len))/ seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
    + head_id * size_per_head + threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template<>
  __global__
void transpose(half* src, half* dst,
    const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int batch_id = tid / (head_num * seq_len * size_per_head);
  int head_id = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
  int seq_id = (tid % (seq_len * size_per_head)) / size_per_head;
  int id = tid % size_per_head;

  int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);
  half2* src_ptr = (half2*)src;
  half2* dst_ptr = (half2*)dst;

  dst_ptr[target_id] = src_ptr[tid];
}


template<typename T>
__global__
void transpose_rebuild_padding(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head,
  const int* mask_offset)
{
  // TODO: optimize this kernel? 
  // do remove_sequence_length_padding
  const int tid = threadIdx.x; // batch * seq_len or valid_word_num
  const int bid = blockIdx.x; // head_num * size_per_head

  const int src_batch_id = (bid + mask_offset[bid]) / seq_len;
  const int src_seq_id = (bid + mask_offset[bid]) % seq_len;

  const int dst_seq_id = bid;

  const int head_id = tid / size_per_head;
  const int hidden_id = tid % size_per_head;
  dst[dst_seq_id * head_num * size_per_head + tid] = src[ src_batch_id * head_num * seq_len * size_per_head +
    head_id * seq_len * size_per_head + src_seq_id * size_per_head + hidden_id];
}

template<typename T>
__global__ void rebuild_sequence_length_padding(const T* src, T* tgt,
                                            const int* mask_offset,
                                            const int n)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int tgt_seq_id = bid + mask_offset[bid];
  const int src_seq_id = bid;

  for(int i = tid; i < n; i += blockDim.x)
  {
    tgt[tgt_seq_id * n + i] = src[src_seq_id * n + i];
  }
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t cublas_handle,
      hipblasLtHandle_t cublaslt_handle,
      DataType_* Q,
      const DataType_* bias_Q,
      DataType_* K,
      const DataType_* bias_K,
      DataType_* V,
      const DataType_* bias_V,
      const DataType_* attr_mask,
      DataType_* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const int int8_mode_,
      const DataType_ scalar)
{
    const int k = head_num * size_per_head;

    dim3 grid;
    dim3 block;

    
    if (int8_mode_ != 0)
    {
      //var for int8
      const float*Qbias_amax_ptr, *Kbias_amax_ptr, *Vbias_amax_ptr, *bmm1_amax_ptr, *Softmax_amax_ptr, *bmm2_amax_ptr, *in_amax_ptr, *Q_aftergemm_amax_ptr, *K_aftergemm_amax_ptr, *V_aftergemm_amax_ptr;
      Qbias_amax_ptr = param_.amaxList + 8;
      Kbias_amax_ptr = param_.amaxList + 16;
      Vbias_amax_ptr = param_.amaxList + 24;
      Softmax_amax_ptr = param_.amaxList + 32;
      bmm2_amax_ptr = param_.amaxList + 36;
      Q_aftergemm_amax_ptr = param_.amaxList + 4;
      K_aftergemm_amax_ptr = param_.amaxList + 12;
      V_aftergemm_amax_ptr = param_.amaxList + 20;
      bmm1_amax_ptr = param_.amaxList + 28;
      in_amax_ptr = param_.amaxList;

      assert(seq_len % COL32_ == 0 && size_per_head%COL32_ == 0);

      if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len){
        if (int8_mode_ == 1)
        {
          add_QK_bias_transform<<<dim3(batch_size*seq_len*2), dim3((head_num * size_per_head)/4), 0, stream>>>((int8_t*)q_buf_, (int8_t*)k_buf_, (const int32_t*) Q, bias_Q, (const int32_t*) K, 
                    bias_K, batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, 
                    seq_len*size_per_head, query_weight_amax_list, in_amax_ptr+2, key_weight_amax_list, 
                    in_amax_ptr+2, Qbias_amax_ptr+3, Kbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
          add_V_bias_transform<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>((int8_t*)v_buf_, (const int32_t *)V, bias_V, batch_size, seq_len, 
                    head_num, size_per_head, seq_len*size_per_head, value_weight_amax_list, 
                    in_amax_ptr+2, Vbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        }
        else
        {
          add_QK_bias_transform<<<dim3(batch_size*seq_len*2), dim3((head_num * size_per_head)/4), 0, stream>>>((int8_t*)q_buf_, (int8_t*)k_buf_, (const int8_t*) Q, bias_Q, (const int8_t*) K, 
                         bias_K, batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, 
                         seq_len*size_per_head, Q_aftergemm_amax_ptr+1, K_aftergemm_amax_ptr+1, 
			 Qbias_amax_ptr+3, Kbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
          add_V_bias_transform<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>((int8_t*)v_buf_, (const int8_t *)V, bias_V, batch_size, seq_len, 
                              head_num, size_per_head, seq_len*size_per_head,
                              V_aftergemm_amax_ptr+1, Vbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        }
      }
      else{
        hipMemset(sequence_id_map_, -1, batch_size * seq_len * sizeof(int));
        mappingRemovePaddingData<<<dim3((param_.valid_word_num + 63)/64), dim3(64)>>>(sequence_id_map_, param_.sequence_id_offset, param_.valid_word_num);
        if (int8_mode_ == 1)
        {
          add_QK_bias_transform_rebuild_padding<<<dim3(param_.valid_word_num*2), dim3((head_num * size_per_head)/4), 0, stream>>>((int8_t*)q_buf_, (int8_t*)k_buf_, (const int32_t*) Q, bias_Q, 
                                          (const int32_t*) K, bias_K, param_.sequence_id_offset, param_.valid_word_num, 
                                          batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, seq_len*size_per_head, 
                                          query_weight_amax_list, in_amax_ptr+2, key_weight_amax_list, in_amax_ptr+2, 
                                          Qbias_amax_ptr+3, Kbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        
          add_V_bias_transform_rebuild_padding<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>((int8_t*)v_buf_, (const int32_t *)V, bias_V, sequence_id_map_, 
                                    param_.valid_word_num, batch_size, seq_len, head_num, 
                                    size_per_head, seq_len*size_per_head, value_weight_amax_list, 
                                    in_amax_ptr+2, Vbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        }
        else
        {
          add_QK_bias_transform_rebuild_padding<<<dim3(param_.valid_word_num*2), dim3((head_num * size_per_head)/4), 0, stream>>>((int8_t*)q_buf_, (int8_t*)k_buf_, (const int8_t*) Q, bias_Q, 
                                          (const int8_t*) K, bias_K, param_.sequence_id_offset, param_.valid_word_num, 
                                          batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, seq_len*size_per_head, 
                                          Q_aftergemm_amax_ptr+1, K_aftergemm_amax_ptr+1, 
                                          Qbias_amax_ptr+3, Kbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        
          add_V_bias_transform_rebuild_padding<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>((int8_t*)v_buf_, (const int8_t *)V, bias_V, sequence_id_map_, 
                                    param_.valid_word_num, batch_size, seq_len, head_num, 
                                    size_per_head, seq_len*size_per_head, 
                                    V_aftergemm_amax_ptr+1, Vbias_amax_ptr+3, use_ORDER_COL32_2R_4R4_);
        }
      }
     
      int batchCount = batch_size * head_num;
      grid.x = seq_len;
      grid.y = batch_size;
      grid.z = head_num;
      
      if (int8_mode_ == 1)
      {     
        cublasLtMM_withAlgo(qk_int_buf_, batchCount, seq_len, seq_len, size_per_head, 
                            size_per_head*seq_len, size_per_head*seq_len, seq_len*seq_len, 
                            (int8_t*)q_buf_, (int8_t*)k_buf_, cublaslt_handle, stream, cublasLtAlgoMap_, use_ORDER_COL32_2R_4R4_, true);

        if (seq_len <= 32){
          if (batch_size * head_num > 960)
            grid.x = ceil(float(seq_len)/32.0f);
          block.x = (seq_len + 31)/32*32;
          softmax_COL32_LE32<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, qk_int_buf_, attr_mask, batch_size, head_num, 
                                                         seq_len, float(scalar), Qbias_amax_ptr + 1, Kbias_amax_ptr + 1, 
                                                         Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        else if (seq_len <= 64){
          assert(seq_len % 2 == 0);
          block.x = (seq_len/2 + 31)/32*32;
          if (batch_size * head_num > 960)
            grid.x = ceil(float(seq_len)/32.0f);
          softmax_COL32_LE64<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, qk_int_buf_, attr_mask, batch_size, head_num, 
                                                         seq_len, float(scalar), Qbias_amax_ptr + 1, Kbias_amax_ptr + 1, 
                                                         Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        else
        {
          assert(seq_len % 4 == 0);
          block.x = (seq_len/4 + 31)/32*32;
          softmax_COL32<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, qk_int_buf_, attr_mask, batch_size, head_num, 
                                                    seq_len, float(scalar), Qbias_amax_ptr + 1, Kbias_amax_ptr + 1, 
                                                    Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        
        cublasLtMM_withAlgo(transpose_dst_int_buf_, batchCount, seq_len, size_per_head, seq_len, 
                            seq_len*seq_len, size_per_head*seq_len, size_per_head*seq_len, (int8_t*)qk_buf_, 
                            (int8_t*)v_buf_, cublaslt_handle, stream, cublasLtAlgoMap_, use_ORDER_COL32_2R_4R4_, true);
    
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          transpose_COL32_kernelLauncher((int8_t*)dst, (const int*)transpose_dst_int_buf_, batch_size, seq_len, head_num, 
                                         size_per_head, Vbias_amax_ptr+1, Softmax_amax_ptr+1, bmm2_amax_ptr+3, stream);
        }
        else
        {
          transpose_COL32_rebuild_padding_kernelLauncher((int8_t*)dst, (const int*)transpose_dst_int_buf_, sequence_id_map_, 
                                                         param_.valid_word_num, batch_size, seq_len, head_num, size_per_head, 
                                                         Vbias_amax_ptr+1, Softmax_amax_ptr+1, bmm2_amax_ptr+3, stream);     
        }
        
      }
      else
      {
        cublasLtMM_withAlgo_int8IO((int8_t*)qk_int_buf_, batchCount, seq_len, seq_len, size_per_head, 
                                   size_per_head*seq_len, size_per_head*seq_len, seq_len*seq_len, 
                                   param_.int8O_gemm_deQ_scale_list[3],
                                   (int8_t*)q_buf_, (int8_t*)k_buf_, cublaslt_handle, stream, cublasLtAlgoMap_, use_ORDER_COL32_2R_4R4_, true);
                         
        if (seq_len <= 32){
          if (batch_size * head_num > 960)
            grid.x = ceil(float(seq_len)/32.0f);
          block.x = (seq_len + 31)/32*32;
          softmax_COL32_LE32<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, (int8_t*)qk_int_buf_, attr_mask, batch_size, head_num, 
                                                         seq_len, float(scalar), bmm1_amax_ptr + 1, 
                                                         Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        else if (seq_len <= 64){
          assert(seq_len % 2 == 0);
          block.x = (seq_len/2 + 31)/32*32;
          if (batch_size * head_num > 960)
            grid.x = ceil(float(seq_len)/32.0f);
          softmax_COL32_LE64<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, (int8_t*)qk_int_buf_, attr_mask, batch_size, head_num, 
                                                         seq_len, float(scalar), bmm1_amax_ptr + 1,  
                                                         Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        else
        {
          assert(seq_len % 4 == 0);
          block.x = (seq_len/4 + 31)/32*32;
          softmax_COL32<<<grid, block, 0, stream>>>((int8_t*)qk_buf_, (int8_t*)qk_int_buf_, attr_mask, batch_size, head_num, 
                                                    seq_len, float(scalar), bmm1_amax_ptr + 1,  
                                                    Softmax_amax_ptr, seq_len*head_num, seq_len*seq_len);
        }
        
        cublasLtMM_withAlgo_int8IO((int8_t*)transpose_dst_int_buf_, batchCount, seq_len, size_per_head, seq_len, 
                                   seq_len*seq_len, size_per_head*seq_len, size_per_head*seq_len, param_.int8O_gemm_deQ_scale_list[4], (int8_t*)qk_buf_, 
                                   (int8_t*)v_buf_, cublaslt_handle, stream, cublasLtAlgoMap_, use_ORDER_COL32_2R_4R4_, true);
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          transpose_COL32_kernelLauncher((int8_t*)dst, (const int8_t*)transpose_dst_int_buf_, batch_size, seq_len, head_num, 
                                         size_per_head, bmm2_amax_ptr+1, bmm2_amax_ptr+3, stream);
        }
        else
        {
          transpose_COL32_rebuild_padding_kernelLauncher((int8_t*)dst, (const int8_t*)transpose_dst_int_buf_, sequence_id_map_, 
                                                         param_.valid_word_num, batch_size, seq_len, head_num, size_per_head, 
                                                         bmm2_amax_ptr+1, 
                                                         bmm2_amax_ptr+3, stream);
        }
      }
    }
    //FP32/FP16
    else{
      if(OpType_ == OperationType::FP32)
      {
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          const int m = batch_size * seq_len;
          const int word_per_block = 1;
          assert(k <= 1024);
          assert(m / word_per_block * 3 <= 65536);
  
          dim3 grid(m / word_per_block * 3);
          dim3 block(k);
          add_QKV_bias<DataType_><<<grid, block, 0, stream>>>(Q, bias_Q, K, bias_K, V, bias_V, q_buf_, k_buf_, v_buf_,
            batch_size, seq_len, head_num, size_per_head, word_per_block);
        }
        else
        {
          add_QKV_bias_rebuild_padding<DataType_><<<param_.valid_word_num, k, 0, stream>>>(Q, bias_Q, K, bias_K, 
            V, bias_V, q_buf_, k_buf_, v_buf_, 
            batch_size, seq_len, head_num, size_per_head, param_.sequence_id_offset);
        }
      }
      else
      {
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          const int word_per_block = 1;
          grid.x = batch_size * seq_len / word_per_block;
          block.x = head_num * size_per_head * word_per_block / 2;
  
          assert(block.x <= 1024);
  
          add_QKV_bias<DataType_><<<grid, block, 0, stream>>>(Q, bias_Q, K, bias_K, V, bias_V, q_buf_, k_buf_, 
          v_buf_, batch_size, seq_len, head_num, size_per_head / 2, word_per_block);
        }
        else
        {
          add_QKV_bias_rebuild_padding<half2><<<param_.valid_word_num, k / 2, 0, stream>>>((half2*)Q, (const half2*)bias_Q, 
            (half2*)K, (const half2*)bias_K, (half2*)V, (const half2*)bias_V, 
            (half2*)q_buf_, (half2*)k_buf_, (half2*)v_buf_,
            batch_size, seq_len, head_num, size_per_head / 2, param_.sequence_id_offset);
        }
      }

      DataType_ alpha = (DataType_)1.0f, beta = (DataType_)0.0f;
      
      check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        seq_len, seq_len, size_per_head,
        &alpha,
        k_buf_, AType_, size_per_head, seq_len * size_per_head,
        q_buf_, BType_, size_per_head, seq_len * size_per_head,
        &beta,
        qk_buf_, CType_, seq_len, seq_len * seq_len,
        batch_size * head_num,
        computeType_,
        static_cast<hipblasGemmAlgo_t>(cublasAlgo_[1])));
        
      //deal with odd seq_len
      if (seq_len % 2 != 0){
        if(seq_len <= 32)
          block.x = 32;
        else if(seq_len > 32 && seq_len <= 64)
          block.x = 64;
        else if(seq_len > 64 && seq_len <= 128)
          block.x = 128;
        else if(seq_len > 128 && seq_len <= 256)
          block.x = 256;
        else if(seq_len > 256 && seq_len <= 512)
          block.x = 512;
        else
          block.x = 1024;

        if(batch_size * head_num <= 120)
        {
          grid.x = batch_size * head_num * seq_len;
          softmax_kernel_v2<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scalar);
        }
        else
        {
          grid.x = batch_size * head_num;
          softmax_kernel<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scalar);
        }
      }
      //deal with even seq_len 
      else{
        grid.x = seq_len;
        if (batch_size * head_num > 360)
          grid.x = ceil(float(seq_len)/32.0f);
        grid.y = batch_size;
        grid.z = head_num;
        if (seq_len <= 32){
          block.x = 32;
          softmax_kernel_v3_LE32<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scalar);
        }
        else{
          if (OpType_ == OperationType::FP16){
            block.x = (seq_len/2 + 31)/32*32;
            softmax_kernel_v3<<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scalar);
          }
          else{
            block.x = (seq_len + 31)/32*32;
            softmax_kernel_v3<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scalar);
          }
        }
        grid.x = grid.y = grid.z = 1;
      }

      check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        size_per_head, seq_len, seq_len,
        &alpha,
        v_buf_, AType_, size_per_head, seq_len * size_per_head,
        qk_buf_, BType_, seq_len, seq_len * seq_len,
        &beta,
        transpose_dst_, CType_, size_per_head, seq_len * size_per_head,
        batch_size * head_num,
        computeType_,
        static_cast<hipblasGemmAlgo_t>(cublasAlgo_[2])));
        
      /* for half2 only */
      if(OpType_ == OperationType::FP16)
      {
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          const int seq_per_block = 4;
          grid.x = batch_size * head_num * seq_len / seq_per_block;
          block.x = seq_per_block * size_per_head / 2;
    
          assert(grid.x * seq_per_block == batch_size * head_num * seq_len);
    
          transpose<DataType_><<<grid, block, 0, stream>>>(transpose_dst_, dst, 
              batch_size, seq_len, head_num, size_per_head / 2);
        }
        else
        {
          transpose_rebuild_padding<half2><<<param_.valid_word_num, k / 2, 0, stream>>>(
            (half2*)transpose_dst_, (half2*)dst, 
            batch_size, seq_len, head_num, size_per_head / 2, param_.sequence_id_offset);
        }
      }
      else
      {
        if(param_.sequence_id_offset == nullptr || param_.valid_word_num == batch_size * seq_len)
        {
          const int seq_per_block = 1;
          grid.x = batch_size * head_num * seq_len / seq_per_block;
          block.x = seq_per_block * size_per_head;
          transpose<DataType_><<<grid, block, 0, stream>>>(transpose_dst_, dst, 
            batch_size, seq_len, head_num, size_per_head);
        }
        else
        {
          transpose_rebuild_padding<DataType_><<<param_.valid_word_num, k, 0, stream>>>(transpose_dst_, dst, 
            batch_size, seq_len, head_num, size_per_head, param_.sequence_id_offset);
        }
      }
    }
}

template void OpenMultiHeadAttention<OperationType::FP32>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t handle,
      hipblasLtHandle_t cublaslt_handle,
      float* Q,
      const float* bias_Q,
      float* K,
      const float* bias_K,
      float* V,
      const float* bias_V,
      const float* attr_mask,
      float* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const int int8_mode_,
      const float scalar);

template void OpenMultiHeadAttention<OperationType::FP16>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t handle,
      hipblasLtHandle_t cublaslt_handle,
      half* Q,
      const half* bias_Q,
      half* K,
      const half* bias_K,
      half* V,
      const half* bias_V,
      const half* attr_mask,
      half* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const int int8_mode_,
      const half scalar);

template void OpenMultiHeadAttention<OperationType::FP32>::trt_add_QKV_bias_kernelLauncher(
  const float* bias_Q,
  const float* bias_K,
  const float* bias_V);

template void OpenMultiHeadAttention<OperationType::FP16>::trt_add_QKV_bias_kernelLauncher(
  const half* bias_Q,
  const half* bias_K,
  const half* bias_V);

template void OpenMultiHeadAttention<OperationType::FP32>::fused_multiHeadAttr_kernelLauncher();
template void OpenMultiHeadAttention<OperationType::FP16>::fused_multiHeadAttr_kernelLauncher();

__global__
void trt_add_QKV_bias_2(const half2* Q, const half2* bias_Q, 
                        const half2* K, const half2* bias_K, 
                        const half2* V, const half2* bias_V, 
                        half2* qkv_buf_,  
                        const int valid_word_num, 
                        const int head_num, const int size_per_head)
{
  // Add bias, and then transpose from 
  // [3, valid_word_num, head, size] -> [valid_word_num, head, 3, size]

  const int seq_id = blockIdx.x;
  const int size_id = threadIdx.x % size_per_head;
  const int head_id = (threadIdx.x - size_id) / size_per_head;

  const int target_offset = blockIdx.x * head_num * 3 * size_per_head + head_id * 3 * size_per_head;

  qkv_buf_[ target_offset + 
          0 * size_per_head +
          size_id] = Q[ seq_id * blockDim.x + threadIdx.x] + bias_Q[threadIdx.x];

  qkv_buf_[ target_offset + 
          1 * size_per_head +
          size_id] = K[ seq_id * blockDim.x + threadIdx.x] + bias_K[threadIdx.x];

  qkv_buf_[ target_offset + 
          2 * size_per_head +
          size_id] = V[ seq_id * blockDim.x + threadIdx.x] + bias_V[threadIdx.x];
}

void add_QKV_bias_transpose_kernelLauncher(
  const half* query_buf, const half* bias_Q,
  const half* key_buf, const half* bias_K,
  const half* value_buf, const half* bias_V,
  half* context_buf, 
  const int valid_word_num, 
  const int head_num, const int size_per_head,
  hipStream_t stream)
{
  dim3 grid;
  dim3 block;
  
  grid.x = 3 * valid_word_num;
  block.x = head_num * size_per_head / 2;
  
  assert(block.x <= 1024);

  trt_add_QKV_bias_2<<<grid, block, 0, stream>>>( (const half2*)query_buf, (const half2*)bias_Q, 
                                                  (const half2*)key_buf, (const half2*)bias_K, 
                                                  (const half2*)value_buf, (const half2*)bias_V, 
                                                  (half2*)context_buf, 
                                                  valid_word_num, 
                                                  head_num, size_per_head / 2);
}


}//namespace cuda
}//namespace fastertransformer

