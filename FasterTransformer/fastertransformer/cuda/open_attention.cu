#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/**
 * Open sourced multi-head attention
 **/

#include "fastertransformer/allocator.h"
#include "fastertransformer/cuda/multi_head_attention.h"
#include "fastertransformer/cuda/open_attention.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
namespace fastertransformer{
namespace cuda{

/**
 * Multi-head attetion open sourced
 */
#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);
                              
  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : 0;
  val = warpReduceMax(val);

  return val;
}

  __inline__ __device__
int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
  return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template<typename T>
__global__
void add_QKV_bias(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{

  T* data_ptr;
  T* buf_ptr;
  const T* bias_ptr;
  
  int m = batch_size * seq_len;
  int n = head_num * size_per_head;

  int qkv_id = blockIdx.x * word_per_block / m;
  int row_offset = (blockIdx.x * word_per_block % m) * n;

  if(qkv_id == 0)
  {
    data_ptr = Q + row_offset;
    buf_ptr = q_buf_;
    bias_ptr = bias_Q;
  }
  else if(qkv_id == 1)
  {
    data_ptr = K + row_offset;
    buf_ptr = k_buf_;
    bias_ptr = bias_K;
  }
  else
  {
    data_ptr = V + row_offset;
    buf_ptr = v_buf_;
    bias_ptr = bias_V;
  }

  int batch_id = (blockIdx.x * word_per_block % m) / seq_len;
  int head_id = threadIdx.x / size_per_head;
  int id_in_head = threadIdx.x % size_per_head;
  int word_start_id = (blockIdx.x * word_per_block) % seq_len;

  T bias = __ldg(&bias_ptr[threadIdx.x]);

  for(int i = word_start_id; i < word_start_id + word_per_block; ++i)
  {
    T tmp = data_ptr[threadIdx.x] + bias;

    int target_id = batch_id * (seq_len * head_num * size_per_head) + head_id * seq_len * size_per_head + 
      i * size_per_head + id_in_head;

    buf_ptr[target_id] = tmp;
    data_ptr += n;
  }
}

template <>
__global__
void add_QKV_bias(__half* Q, const __half* bias_Q, __half* K, const __half* bias_K, __half* V, const __half* bias_V, 
  __half* q_buf_, __half* k_buf_, __half* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_id = tid / (head_num * seq_len * size_per_head);
  int seq_id = (tid % (head_num * seq_len * size_per_head)) / (head_num * size_per_head);
  int head_id = (tid % (head_num * size_per_head)) / size_per_head;
  int id = tid % size_per_head;
  int target_id = target_index(batch_id, seq_id, head_id, id, batch_size, seq_len, head_num, size_per_head);

  int bias_id = threadIdx.x;

  half2* src_ptr = (half2*)Q;
  half2* dst_ptr = (half2*)q_buf_;
  const half2* bias_ptr = (const half2*)bias_Q;

  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)K;
  dst_ptr = (half2*)k_buf_;
  bias_ptr = (const half2*)bias_K;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)V;
  dst_ptr = (half2*)v_buf_;
  bias_ptr = (const half2*)bias_V;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));
}

template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, 
  const T scaler)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;
    int mask_offset = batch_id * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
      mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val): -1e-20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
      mask_offset += seq_len;
    }
}


template <typename T>
__global__
void softmax_kernel_v2(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, 
  const int seq_len, const float scaler)
{
    int batch_id = blockIdx.x / head_num / seq_len;
    int seq_id = blockIdx.x % seq_len;
    int qk_offset = blockIdx.x * seq_len;
    int mask_offset = batch_id * seq_len * seq_len + seq_id * seq_len;

    __shared__ float s_sum, s_max;

    float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
    mask_val = (1.0f - mask_val) * -10000.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val) : -1e-20f;
    float max_val = blockReduceMax<float>(tmp);
    if(threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);

    if(threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if(threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

template<typename T>
__global__
void transpose(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len))/ seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
    + head_id * size_per_head + threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template<>
  __global__
void transpose(__half* src, __half* dst,
    const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int batch_id = tid / (head_num * seq_len * size_per_head);
  int head_id = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
  int seq_id = (tid % (seq_len * size_per_head)) / size_per_head;
  int id = tid % size_per_head;

  int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);
  half2* src_ptr = (half2*)src;
  half2* dst_ptr = (half2*)dst;

  dst_ptr[target_id] = src_ptr[tid];
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t cublas_handle,
      DataType_* Q,
      const DataType_* bias_Q,
      DataType_* K,
      const DataType_* bias_K,
      DataType_* V,
      const DataType_* bias_V,
      const DataType_* attr_mask,
      DataType_* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const DataType_ scaler)
{

    int m = batch_size * seq_len;
    int k = head_num * size_per_head;

    dim3 grid;
    dim3 block;

    if(OpType_ == OperationType::FP32)
    {
//      const int word_per_block = 32;
      const int word_per_block = 1;
      assert(k > 1024);
      assert(m / word_per_block * 3 > 65536);

      dim3 grid(m / word_per_block * 3);
      dim3 block(k);
      add_QKV_bias<DataType_><<<grid, block, 0, stream>>>(Q, bias_Q, K, bias_K, V, bias_V, q_buf_, k_buf_, v_buf_,
          batch_size, seq_len, head_num, size_per_head, word_per_block);
    }
    else
    {
      const int word_per_block = 1;
      grid.x = batch_size * seq_len / word_per_block;
      block.x = head_num * size_per_head * word_per_block / 2;

      assert(block.x);

      add_QKV_bias<DataType_><<<grid, block, 0, stream>>>(Q, bias_Q, K, bias_K, V, bias_V, q_buf_, k_buf_, 
      v_buf_, batch_size, seq_len, head_num, size_per_head / 2, word_per_block);
    }

    DataType_ alpha = (DataType_)1.0f, beta = (DataType_)0.0f;
    
    check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
      HIPBLAS_OP_T, HIPBLAS_OP_N,
      seq_len, seq_len, size_per_head,
      &alpha,
      k_buf_, AType_, size_per_head, seq_len * size_per_head,
      q_buf_, BType_, size_per_head, seq_len * size_per_head,
      &beta,
      qk_buf_, CType_, seq_len, seq_len * seq_len,
      batch_size * head_num,
      computeType_,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo_[1])));

    if(seq_len <= 32)
      block.x = 32;
    else if(seq_len > 32 && seq_len <= 64)
      block.x = 64;
    else if(seq_len > 64 && seq_len <= 128)
      block.x = 128;
    else if(seq_len > 128 && seq_len <= 256)
      block.x = 256;
    else if(seq_len > 256 && seq_len <= 512)
      block.x = 512;
    else
      block.x = 1024;

    if(batch_size * head_num <= 120)
    {
      grid.x = batch_size * head_num * seq_len;
      softmax_kernel_v2<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scaler); 
    }
    else
    {
      grid.x = batch_size * head_num;
      softmax_kernel<DataType_><<<grid, block, 0, stream>>>(qk_buf_, attr_mask, batch_size, head_num, seq_len, scaler); 
    }

    check_cuda_error(hipblasGemmStridedBatchedEx(cublas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      size_per_head, seq_len, seq_len,
      &alpha,
      v_buf_, AType_, size_per_head, seq_len * size_per_head,
      qk_buf_, BType_, seq_len, seq_len * seq_len,
      &beta,
      transpose_dst_, CType_, size_per_head, seq_len * size_per_head,
      batch_size * head_num,
      computeType_,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo_[2])));

/* for half2 only */
    if(OpType_ == OperationType::HALF)
    {
      const int seq_per_block = 4;
  //    const int seq_per_block = 1;
      grid.x = batch_size * head_num * seq_len / seq_per_block;
      block.x = seq_per_block * size_per_head / 2;

      assert(grid.x * seq_per_block != batch_size * head_num * seq_len);

      transpose<DataType_><<<grid, block, 0, stream>>>(transpose_dst_, dst, 
          batch_size, seq_len, head_num, size_per_head / 2);
    }
    else
    {
      const int seq_per_block = 1;
      grid.x = batch_size * head_num * seq_len / seq_per_block;
      block.x = seq_per_block * size_per_head;
      transpose<DataType_><<<grid, block, 0, stream>>>(transpose_dst_, dst, 
          batch_size, seq_len, head_num, size_per_head);
    }
}

template void OpenMultiHeadAttention<OperationType::FP32>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t handle,
      float* Q,
      const float* bias_Q,
      float* K,
      const float* bias_K,
      float* V,
      const float* bias_V,
      const float* attr_mask,
      float* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const float scaler);

template void OpenMultiHeadAttention<OperationType::HALF>::multiHeadAttr_nofuse_kernelLauncher(
      hipStream_t stream,
      hipblasHandle_t handle,
      __half* Q,
      const __half* bias_Q,
      __half* K,
      const __half* bias_K,
      __half* V,
      const __half* bias_V,
      const __half* attr_mask,
      __half* dst,
      const int batch_size,
      const int seq_len,
      const int head_num,
      const int size_per_head,
      const __half scaler);
}//namespace cuda
}//namespace fastertransformer
