#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

std::tuple<at::Tensor, at::Tensor> multi_tensor_l2norm_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::optional<bool> per_tensor_python,
  at::Tensor found_inf,
  at::Tensor inv_scale);

template<typename T>
__device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T1, typename T2>
__device__ __forceinline__ void load_store_with_cast(T1* dst, T2* src, int dst_offset, int src_offset) {
  for (size_t i = 0; i < ILP; ++i) {
    dst[dst_offset + i] = static_cast<T1>(src[src_offset + i]);
  }
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

typedef enum{
  MOMENT_MODE_0   =0, // L2 regularization mode
  MOMENT_MODE_1   =1  // Decoupled weight decay mode
} adamMode_t;

using MATH_T = float;

template<typename grad_t, typename param_t>
struct LAMBStage1Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<4>& tl,
    const float beta1,
    const float beta2,
    const float beta3,
    const float beta1_correction,
    const float beta2_correction,
    const float epsilon,
    adamMode_t mode,
    const float decay,
    const float* global_grad_norm,
    const float max_global_grad_norm,
    const float* found_inf,
    const float* inv_scale)
  {
    if (*found_inf) {
      return;
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float clipped_global_grad_norm = (*global_grad_norm) > max_global_grad_norm ? (*global_grad_norm) / max_global_grad_norm : 1.0f;

    grad_t* g = (grad_t*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    param_t* p = (param_t*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    param_t* m = (param_t*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    param_t* v = (param_t*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    MATH_T r_g[ILP];
    MATH_T r_p[ILP];
    MATH_T r_m[ILP];
    MATH_T r_v[ILP];
    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 &&
       chunk_size % ILP == 0 &&
       is_aligned(g) &&
       is_aligned(p) &&
       is_aligned(m) &&
       is_aligned(v))
    {
      grad_t l_g[ILP];
      param_t l_p[ILP];
      param_t l_m[ILP];
      param_t l_v[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(l_g, g, 0, i_start);
        if (decay != 0)
          load_store(l_p, p, 0, i_start);
        load_store(l_m, m, 0, i_start);
        load_store(l_v, v, 0, i_start);
        // unpack
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_g[ii] = l_g[ii] * (*inv_scale);
          if (decay == 0) {
            r_p[ii] = MATH_T(0);
          }
          else {
            r_p[ii] = l_p[ii];
          }
          r_m[ii] = l_m[ii];
          r_v[ii] = l_v[ii];
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          l_p[ii] = r_p[ii];
          l_m[ii] = r_m[ii];
          l_v[ii] = r_v[ii];
        }
        // store
        load_store_with_cast<grad_t, MATH_T>(g, l_p, i_start, 0);
        load_store(m, l_m, i_start, 0);
        load_store(v, l_v, i_start, 0);
      }
    }
    else
    {
      // see note in multi_tensor_scale_kernel.cu
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_g[ILP];
        MATH_T r_p[ILP];
        MATH_T r_m[ILP];
        MATH_T r_v[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_g[ii] = g[i];
            // special ?optimization? for lamb stage 1
            if (decay == 0) {
              r_p[ii] = MATH_T(0);
            }
            else {
              r_p[ii] = p[i];
            }
            r_m[ii] = m[i];
            r_v[ii] = v[i];
          } else {
            r_g[ii] = MATH_T(0);
            r_p[ii] = MATH_T(0);
            r_m[ii] = MATH_T(0);
            r_v[ii] = MATH_T(0);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            g[i] = r_p[ii];
            m[i] = r_m[ii];
            v[i] = r_v[ii];
          }
        }
      }
    }
  }
};

// Step 2 reads in 'update' value and per-tensor param_norm and update_norm.
// It computes new parameter value.
template<typename T, typename master_param_t>
struct LAMBStage2Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    const float* per_tensor_param_norm,
    const float* per_tensor_update_norm,
    const float learning_rate,
    const float decay,
    bool use_nvlamb,
    float* found_inf,
    float* inv_scale)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;
    if (*found_inf) {
      return;
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    MATH_T ratio = learning_rate;
    // nvlamb: apply adaptive learning rate to all parameters
    // otherwise, only apply to those with non-zero weight decay
    if (use_nvlamb || (decay != 0.0))
    {
      float param_norm = per_tensor_param_norm[tensor_num];
      float update_norm = per_tensor_update_norm[tensor_num];
      ratio = (update_norm != 0.0f && param_norm != 0.0f) ? learning_rate * (param_norm / update_norm) : learning_rate;
    }

    T* update = (T*)tl.addresses[0][tensor_loc];
    update += chunk_idx*chunk_size;

    master_param_t* master_p = (master_param_t*)tl.addresses[1][tensor_loc];
    master_p += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[2][tensor_loc];
    p += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 &&
       chunk_size % ILP == 0 &&
       is_aligned(p) &&
       is_aligned(update))
    {
      T r_p[ILP];
      T r_update[ILP];
      master_param_t r_master_p[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(r_p, p, 0, i_start);
        load_store(r_update, update, 0, i_start);
        load_store(r_master_p, master_p, 0, i_start);
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_master_p[ii] = static_cast<MATH_T>(r_p[ii]) - (ratio * static_cast<MATH_T>(r_update[ii]));
          r_p[ii] = static_cast<T>(r_master_p[ii]);
        }
        load_store(p, r_p, i_start, 0);
        load_store(master_p, r_master_p, i_start, 0);
      }
    }
    else
    {
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_p[ILP];
        MATH_T r_update[ILP];
        MATH_T r_master_p[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_p[ii] = p[i];
            r_update[ii] = update[i];
            r_master_p[ii] = master_p[i];
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_master_p[ii] = r_master_p[ii] - (ratio * r_update[ii]);
          r_p[ii] = r_master_p[ii];
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            master_p[i] = r_master_p[ii];
            p[i] = r_p[ii];
          }
        }
      }
    }
  }
};

void multi_tensor_lamb_out_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  const float lr,
  const float beta1,
  const float beta2,
  const float epsilon,
  const int step,
  const int bias_correction,
  const float weight_decay,
  const int grad_averaging,
  const int mode,
  at::Tensor global_grad_norm,
  const float max_grad_norm,
  at::optional<bool> use_nvlamb_python,
  at::Tensor found_inf,
  at::Tensor inv_scale)
{
  assert(tensor_lists.size() == 5);
  using namespace at;
  // Master weight and 32bit momentum(potentially changing) is not handled by this
  // So we assume every tensor are all in the same type

  bool use_nvlamb = use_nvlamb_python.has_value() ? use_nvlamb_python.value() : false;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  // Handle grad averaging mode
  float beta3 = 1.0f;
  if (grad_averaging == 1) beta3 = 1 - beta1;

  std::vector<std::vector<at::Tensor>> stage1_tensor_lists{
    tensor_lists[0],
    tensor_lists[1],
    tensor_lists[2],
    tensor_lists[3],
  };
  std::vector<std::vector<at::Tensor>> grad_list(tensor_lists.begin(), tensor_lists.begin()+1);
  std::vector<std::vector<at::Tensor>> param_list(tensor_lists.begin()+1, tensor_lists.begin()+2);

  // Compute per tensor param norm
  auto param_norm_tuple = multi_tensor_l2norm_cuda(chunk_size, noop_flag, param_list, true, found_inf, inv_scale);

  // We now in-place modify grad to store update before compute its norm
  // Generally this is not a issue since people modify grad in step() method all the time
  // We can also grab list of empty tensor to avoid this, but I'd like to save space/cpu code
  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
    multi_tensor_apply<4>(
      BLOCK_SIZE,
      chunk_size,
      noop_flag,
      stage1_tensor_lists,
      LAMBStage1Functor<scalar_t_0, float>(),
      beta1,
      beta2,
      beta3, // 1-beta1 or 1 depends on averaging mode
      bias_correction1,
      bias_correction2,
      epsilon,
      (adamMode_t) mode,
      weight_decay,
      global_grad_norm.data_ptr<float>(),
      max_grad_norm,
      found_inf.data_ptr<float>(),
      inv_scale.data_ptr<float>()); )

  // Compute update norms
  auto update_norm_tuple = multi_tensor_l2norm_cuda(chunk_size, noop_flag, grad_list, true, found_inf, inv_scale);

  std::vector<std::vector<at::Tensor>> grad_param_list{ tensor_lists[0], tensor_lists[1], tensor_lists[4]  };
  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_2",
      multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
       	noop_flag,
        grad_param_list,
        LAMBStage2Functor<scalar_t_0, float>(),
        std::get<1>(param_norm_tuple).data_ptr<float>(),
        std::get<1>(update_norm_tuple).data_ptr<float>(),
        lr,
	weight_decay,
	use_nvlamb,
    found_inf.data_ptr<float>(),
    inv_scale.data_ptr<float>()); )

  AT_CUDA_CHECK(hipGetLastError());

}
