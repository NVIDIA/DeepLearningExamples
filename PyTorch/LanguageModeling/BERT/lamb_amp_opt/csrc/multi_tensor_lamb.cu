#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>
#include <iostream>

#include <assert.h>
#include <type_traits>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

std::tuple<at::Tensor, at::Tensor> multi_tensor_l2norm_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::optional<bool> per_tensor_python);

template<typename T>
__device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

typedef enum{
  MOMENT_MODE_0   =0, // L2 regularization mode
  MOMENT_MODE_1   =1  // Decoupled weight decay mode
} adamMode_t;

using MATH_T = float;

#include<cstdio>
template<typename T, typename param_t>
struct LAMBStage1Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<4>& tl,
    const float beta1,
    const float beta2,
    const float beta3,
    const int* step_ptr,
    const int bias_correction,
    const float epsilon,
    adamMode_t mode,
    const float decay,
    const float* global_grad_norm,
    const float* max_global_grad_norm,
    const float* found_inf,
    const float* inv_scale)
  {
    if (*noop_gmem) {
      return;
    }

    float beta1_correction = 1.0f;
    float beta2_correction = 1.0f;
    if (bias_correction == 1) {
      int step = *step_ptr;
      beta1_correction = 1 - std::pow(beta1, step);
      beta2_correction = 1 - std::pow(beta2, step);
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float clipped_global_grad_norm = (*global_grad_norm) > (*max_global_grad_norm) ? (*global_grad_norm) / (*max_global_grad_norm) : 1.0f;

    T* g = (T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    param_t* p = (param_t*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    param_t* m = (param_t*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    param_t* v = (param_t*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    MATH_T r_g[ILP];
    MATH_T r_p[ILP];
    MATH_T r_m[ILP];
    MATH_T r_v[ILP];
    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 &&
       chunk_size % ILP == 0 &&
       is_aligned(g) &&
       is_aligned(p) &&
       is_aligned(m) &&
       is_aligned(v))
    {
      T l_g[ILP];
      param_t l_p[ILP];
      param_t l_m[ILP];
      param_t l_v[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(l_g, g, 0, i_start);
        if (decay != 0)
          load_store(l_p, p, 0, i_start);
        load_store(l_m, m, 0, i_start);
        load_store(l_v, v, 0, i_start);
        // unpack
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_g[ii] = l_g[ii] * (*inv_scale);
          if (decay == 0) {
            r_p[ii] = MATH_T(0);
          }
          else {
            r_p[ii] = l_p[ii];
          }
          r_m[ii] = l_m[ii];
          r_v[ii] = l_v[ii];
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
          // printf("(%d %d) r_g: %f, r_p: %f, r_m: %f, r_v: %f\n", i_start, ii, r_g[ii], r_p[ii], r_m[ii], r_v[ii]);
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          l_p[ii] = r_p[ii];
          // Difference from APEX's LAMB kernel. `g` and `p` can be different dtypes.
          l_g[ii] = r_p[ii];
          l_m[ii] = r_m[ii];
          l_v[ii] = r_v[ii];
        }
        // store
        load_store(g, l_g, i_start, 0);
        load_store(m, l_m, i_start, 0);
        load_store(v, l_v, i_start, 0);
      }
    }
    else
    {
      // see note in multi_tensor_scale_kernel.cu
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_g[ILP];
        MATH_T r_p[ILP];
        MATH_T r_m[ILP];
        MATH_T r_v[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_g[ii] = g[i] * (*inv_scale);
            // special ?optimization? for lamb stage 1
            if (decay == 0) {
              r_p[ii] = MATH_T(0);
            }
            else {
              r_p[ii] = p[i];
            }
            r_m[ii] = m[i];
            r_v[ii] = v[i];
          } else {
            r_g[ii] = MATH_T(0);
            r_p[ii] = MATH_T(0);
            r_m[ii] = MATH_T(0);
            r_v[ii] = MATH_T(0);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            g[i] = r_p[ii];
            m[i] = r_m[ii];
            v[i] = r_v[ii];
          }
        }
      }
    }
  }
};

// Step 2 reads in 'update' value and per-tensor param_norm and update_norm.
// It computes new parameter value.
// N == 2: FP32 params, no master params
// N == 3: FP16 params, FP32 master params.
template<typename T, int N, typename param_t>
struct LAMBStage2Functor
{
  static_assert((N == 2 && std::is_same<T, param_t>::value) || (N == 3 && std::is_same<param_t, float>::value), "");
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<N>& tl,
    const float* per_tensor_param_norm,
    const float* per_tensor_update_norm,
    const float* learning_rate,
    const float decay,
    bool use_nvlamb)
  {
    if (*noop_gmem) {
      return;
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    MATH_T ratio = *learning_rate;
    // nvlamb: apply adaptive learning rate to all parameters
    // otherwise, only apply to those with non-zero weight decay
    if (use_nvlamb || (decay != 0.0))
    {
      float param_norm = per_tensor_param_norm[tensor_num];
      float update_norm = per_tensor_update_norm[tensor_num];
      ratio = (update_norm != 0.0f && param_norm != 0.0f) ? *learning_rate * (param_norm / update_norm) : *learning_rate;
    }

    T* update = (T*)tl.addresses[0][tensor_loc];
    update += chunk_idx*chunk_size;

    param_t* p = (param_t*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* out_p;
    if (N == 3) {
      out_p = (T*)tl.addresses[2][tensor_loc];
      out_p += chunk_idx*chunk_size;
    }

    n -= chunk_idx*chunk_size;

    // to make things simple, we put aligned case in a different code path
    bool can_use_aligned_path = n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(p) && is_aligned(update);
    if (N == 3) {
      can_use_aligned_path = can_use_aligned_path && is_aligned(out_p);
    }
    if(can_use_aligned_path)
    {
      param_t r_p[ILP];
      T r_update[ILP];
      T r_out_p[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(r_p, p, 0, i_start);
        load_store(r_update, update, 0, i_start);
        if (N == 3) {
          load_store(r_out_p, out_p, 0, i_start);
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_p[ii] = static_cast<MATH_T>(r_p[ii]) - (ratio * static_cast<MATH_T>(r_update[ii]));
          if (N == 3) {
            r_out_p[ii] = r_p[ii];
          }
        }
        load_store(p, r_p, i_start, 0);
        if (N == 3) {
          load_store(out_p, r_out_p, i_start, 0);
        }
      }
    }
    else
    {
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_p[ILP];
        MATH_T r_update[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_p[ii] = p[i];
            r_update[ii] = update[i];
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_p[ii] = r_p[ii] - (ratio * r_update[ii]);
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            p[i] = r_p[ii];
            if (N == 3) {
              out_p[i] = p[i];
            }
          }
        }
      }
    }
  }
};


void multi_tensor_lamb_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor lr,
  const float beta1,
  const float beta2,
  const float epsilon,
  const at::Tensor step,
  const int bias_correction,
  const float weight_decay,
  const int grad_averaging,
  const int mode,
  at::Tensor global_grad_norm,
  at::Tensor max_grad_norm,
  at::optional<bool> use_nvlamb_python,
  at::Tensor found_inf,
  at::Tensor inv_scale)
{
  // n_tensors == 5: FP16 model params & FP32 master params
  // n_tensors == 4: FP32 model params & NO FP32 master params
  const auto n_tensors = tensor_lists.size();
  assert(n_tensors == 4 || n_tensors == 5);
  using namespace at;

  bool use_nvlamb = use_nvlamb_python.has_value() ? use_nvlamb_python.value() : false;

  // note(mkozuki): move bias handling below to functor
  // Handle bias correction mode
  // float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  // if (bias_correction == 1) {
  //   bias_correction1 = 1 - std::pow(beta1, step);
  //   bias_correction2 = 1 - std::pow(beta2, step);
  // }

  // Handle grad averaging mode
  float beta3 = 1.0f;
  if (grad_averaging == 1) beta3 = 1 - beta1;

  std::vector<std::vector<at::Tensor>> stage1_tensor_lists(tensor_lists.begin(), tensor_lists.begin() + 4);
  std::vector<std::vector<at::Tensor>> grad_list(tensor_lists.begin(), tensor_lists.begin()+1);
  std::vector<std::vector<at::Tensor>> param_list(tensor_lists.begin()+1, tensor_lists.begin()+2);

  // Compute per tensor param norm
  auto param_norm_tuple = multi_tensor_l2norm_cuda(chunk_size, noop_flag, param_list, true);

  // We now in-place modify grad to store update before compute its norm
  // Generally this is not a issue since people modify grad in step() method all the time
  // We can also grab list of empty tensor to avoid this, but I'd like to save space/cpu code
  if (n_tensors == 4) {
    DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
        multi_tensor_apply<4>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          stage1_tensor_lists,
          LAMBStage1Functor<scalar_t_0, scalar_t_0>(),
          beta1,
          beta2,
          beta3, // 1-beta1 or 1 depends on averaging mode
          // bias_correction1,
          // bias_correction2,
          step.data_ptr<int>(),
          bias_correction,
          epsilon,
          (adamMode_t) mode,
          weight_decay,
          global_grad_norm.data_ptr<float>(),
          max_grad_norm.data_ptr<float>(),
          found_inf.data_ptr<float>(),
          inv_scale.data_ptr<float>()); )
  } else {
    DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
        multi_tensor_apply<4>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          stage1_tensor_lists,
          LAMBStage1Functor<scalar_t_0, float>(),
          beta1,
          beta2,
          beta3, // 1-beta1 or 1 depends on averaging mode
          // bias_correction1,
          // bias_correction2,
          step.data_ptr<int>(),
          bias_correction,
          epsilon,
          (adamMode_t) mode,
          weight_decay,
          global_grad_norm.data_ptr<float>(),
          max_grad_norm.data_ptr<float>(),
          found_inf.data_ptr<float>(),
          inv_scale.data_ptr<float>()); )
  }

  // Compute update norms
  auto update_norm_tuple = multi_tensor_l2norm_cuda(chunk_size, noop_flag, grad_list, true);

  std::vector<std::vector<at::Tensor>> grad_param_list(tensor_lists.begin(), tensor_lists.begin()+2);
  if (n_tensors == 4) {
    DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_2",
        multi_tensor_apply<2>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          grad_param_list,
          LAMBStage2Functor<scalar_t_0, 2, scalar_t_0>(),
          std::get<1>(param_norm_tuple).data_ptr<float>(),
          std::get<1>(update_norm_tuple).data_ptr<float>(),
          lr.data_ptr<float>(),
      weight_decay,
      use_nvlamb); )
  } else {
    grad_param_list.push_back(tensor_lists[4]);
    DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_2",
        multi_tensor_apply<3>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          grad_param_list,
          LAMBStage2Functor<scalar_t_0, 3, float>(),
          std::get<1>(param_norm_tuple).data_ptr<float>(),
          std::get<1>(update_norm_tuple).data_ptr<float>(),
          lr.data_ptr<float>(),
      weight_decay,
      use_nvlamb); )
  }
  AT_CUDA_CHECK(hipGetLastError());

}
