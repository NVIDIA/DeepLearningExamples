#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <mma.h>
#include <cuda_fp16.hpp>

#include <fstream>
#include <iomanip>
#include <iostream>
#include <vector>

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "shared_utils.cuh"

using namespace nvcuda;

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void dotBasedInteractFwdKernelNonAligned(const __half *__restrict input,
                                                                                        __half *__restrict output,
                                                                                        uint batch_size,
                                                                                        uint num_rows,
                                                                                        uint num_cols,
                                                                                        uint num_rows_after_padding,
                                                                                        uint num_cols_after_padding,
                                                                                        uint smem_elems_per_warp,
                                                                                        uint smem_rows_per_warp,
                                                                                        uint output_size,
                                                                                        uint num_row_steps,
                                                                                        uint num_col_steps,
                                                                                        uint padding_size) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2); //each threadblock covers multiple (4) samples
  //num_rows is num of categoricals + 1, num_cols is embedding/bottom_mlp size
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id; //each warp covers a sample
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1); //0...32, within a sample

  extern __shared__ half shmem_dynamic[];
  half *shmem = shmem_dynamic + (warp_id * smem_elems_per_warp);

  //skip to the input for our warp
  const half *sample_input = input + num_rows * num_cols * sample_id;

  //copy all rows of our input (all embeddings and bottom_mlp)
  for (uint i = 0; i < num_rows; ++i, sample_input += num_cols) {
    //each thread is assigned pieces to copy based on lane_id
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      (shmem + i * SMEM_STRIDE)[idx] = sample_input[idx];
    }
  }

  uint idx = lane_id + num_cols;
  //pad each embedding to num_cols_after_padding
  //this assumes that num_cols_after_padding-num_cols<= WARP_SIZE
  if (idx < num_cols_after_padding) {
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = __float2half(0);
    }
  }

  //add more fake embeddings filled with zeros so we can better use cores
  //zero out 4 cells at once, hence the >>2
  half4 zeros;
  zeros.vals[0].x = __float2half(0);
  zeros.vals[0].y = __float2half(0);
  zeros.vals[1].x = __float2half(0);
  zeros.vals[1].y = __float2half(0);
  if (lane_id < (num_cols_after_padding >> 2)) {
    for (int i = num_rows; i < num_rows_after_padding; i++) {
      ((half4 *)(shmem + i * SMEM_STRIDE))[lane_id] = zeros;
    }
  }
  __syncwarp();
  half *gmem_output = output + output_size * sample_id;

  //copy over the bottom_mlp_output into the final result
  //assumes bottom_mlp_output is at the start of the input
  for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
    gmem_output[idx] = shmem[idx];
  }

  //compute the dot product
  wmma::fragment<wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc[M_BLOCKS][M_BLOCKS];

  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      wmma::fill_fragment(acc[i][j], 0);
    }
  }

  for (int k_step = 0; k_step < num_col_steps; k_step++) {
    wmma::fragment<wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::row_major> a[M_BLOCKS];
    wmma::fragment<wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::col_major> b[M_BLOCKS];
    for (int j = 0; j < M_BLOCKS; j++) {
      int base_row = (j < M_BLOCKS - 1) ? j * 16 : smem_rows_per_warp - 16;
      const half *tile_ptr = shmem + (base_row * SMEM_STRIDE + k_step * 16);
      wmma::load_matrix_sync(a[j], tile_ptr, SMEM_STRIDE);
      wmma::load_matrix_sync(b[j], tile_ptr, SMEM_STRIDE);
    }
    for (int i = 0; i < M_BLOCKS; i++) {
      for (int j = 0; j < M_BLOCKS; j++) {
        wmma::mma_sync(acc[i][j], a[i], b[j], acc[i][j]);
      }
    }
  }
  float *shmem_store = reinterpret_cast<float *>(shmem);
  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      float *tile_ptr = shmem_store + (i * 16 * SMEM_STRIDE_ACC + j * 16);
      wmma::store_matrix_sync(tile_ptr, acc[i][j], SMEM_STRIDE_ACC, wmma::mem_row_major);
    }
  }

  // skip over the part where we copied the bottom_mlp_output
  half *gmem_interact_output = gmem_output + num_cols;

  // copy over the dot product result into the output
  int lastRowBlockOffset = M_BLOCKS * 16 - smem_rows_per_warp;
  int srcLine = 0;
  for (int i = 0; i < num_rows; ++i, ++srcLine) {
    if (i == ((M_BLOCKS - 1) * 16)) {
      srcLine += lastRowBlockOffset;
    }
    if (lane_id < i) { //this assumes we have num_categorical_features<WARP_SIZE
      uint offset = (i * (i - 1)) >> 1;
      gmem_interact_output[offset + lane_id] = __float2half(shmem_store[srcLine * SMEM_STRIDE_ACC + lane_id]);
    }
  }
  // Add padding to the output vectors
  if (lane_id < padding_size) {
    gmem_output[output_size - lane_id - 1] = __float2half(0);
  }

}


template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void dotBasedInteractFwdKernel(const __half *__restrict input,
                                                                              __half *__restrict output,
                                                                              uint batch_size,
                                                                              uint num_rows,
                                                                              uint num_cols,
                                                                              uint num_rows_after_padding,
                                                                              uint num_cols_after_padding,
                                                                              uint smem_elems_per_warp,
                                                                              uint smem_rows_per_warp,
                                                                              uint output_size,
                                                                              uint num_row_steps,
                                                                              uint num_col_steps,
                                                                              uint padding_size) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2); //each threadblock covers multiple (4) samples
  //num_rows is num of categoricals + 1, num_cols is embedding/bottom_mlp size
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id; //each warp covers a sample
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1); //0...32, within a sample

  extern __shared__ half shmem_dynamic[];
  half *shmem = shmem_dynamic + (warp_id * smem_elems_per_warp); //piece of threadblocks memory corresponding to our sample

  const half *sample_input = input + num_rows * num_cols * sample_id; //jump to our sample
  //loop over embeddings, and copy each into shmem (but assume size is <=128>)
  if (lane_id < (num_cols >> 2)) {//divided by 4 because we copy four at once
    for (int i = 0; i < num_rows; ++i, sample_input += num_cols) {
      ((float2 *)(shmem + i * SMEM_STRIDE))[lane_id] = ((float2 *)sample_input)[lane_id];
    }
  }

  //pad each embedding to num_cols_after_padding
  //this assumes that num_cols_after_padding-num_cols<= WARP_SIZE
  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {// the padding is to compute in tiles
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = __float2half(0);
    }
  }

  //add more fake embeddings filled with zeros so we can better use cores
  //zero out 4 cells at once, hence the >>2
  half4 zeros;
  zeros.vals[0].x = __float2half(0);
  zeros.vals[0].y = __float2half(0);
  zeros.vals[1].x = __float2half(0);
  zeros.vals[1].y = __float2half(0);
  if (lane_id < (num_cols_after_padding >> 2)) {
    for (int i = num_rows; i < num_rows_after_padding; i++) {
      ((half4 *)(shmem + i * SMEM_STRIDE))[lane_id] = zeros;
    }
  }
  __syncwarp();
  half *gmem_output = output + output_size * sample_id; //copy over bottom mlp into output memory
  if (lane_id < (num_cols >> 2)) {
    ((float2 *)gmem_output)[lane_id] = ((float2 *)shmem)[lane_id];
  }

  //compute the dot product
  wmma::fragment<wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc[M_BLOCKS][M_BLOCKS];

  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      wmma::fill_fragment(acc[i][j], 0);
    }
  }

  for (int k_step = 0; k_step < num_col_steps; k_step++) {
    wmma::fragment<wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::row_major> a[M_BLOCKS];
    wmma::fragment<wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::col_major> b[M_BLOCKS];
    for (int j = 0; j < M_BLOCKS; j++) {
      int base_row = (j < M_BLOCKS - 1) ? j * 16 : smem_rows_per_warp - 16;
      const half *tile_ptr = shmem + (base_row * SMEM_STRIDE + k_step * 16);
      wmma::load_matrix_sync(a[j], tile_ptr, SMEM_STRIDE);
      wmma::load_matrix_sync(b[j], tile_ptr, SMEM_STRIDE);
    }
    for (int i = 0; i < M_BLOCKS; i++) {
      for (int j = 0; j < M_BLOCKS; j++) {
        wmma::mma_sync(acc[i][j], a[i], b[j], acc[i][j]);
      }
    }
  }
  float *shmem_store = reinterpret_cast<float *>(shmem);
  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      float *tile_ptr = shmem_store + (i * 16 * SMEM_STRIDE_ACC + j * 16);
      wmma::store_matrix_sync(tile_ptr, acc[i][j], SMEM_STRIDE_ACC, wmma::mem_row_major);
    }
  }

  // skip over the part where we copied the bottom_mlp_output
  half *gmem_interact_output = gmem_output + num_cols;

  // copy over the dot product result into the output
  int lastRowBlockOffset = M_BLOCKS * 16 - smem_rows_per_warp;
  int srcLine = 0;
  for (int i = 0; i < num_rows; ++i, ++srcLine) {
    if (i == ((M_BLOCKS - 1) * 16)) {
      srcLine += lastRowBlockOffset;
    }
    if (lane_id < i) { //this assumes we have num_categorical_features (num_rows-1)<WARP_SIZE
      uint offset = (i * (i - 1)) >> 1;
      gmem_interact_output[offset + lane_id] = __float2half(shmem_store[srcLine * SMEM_STRIDE_ACC + lane_id]);
    }
  }

  // Add padding to the output vectors
  if (lane_id < padding_size) {
    gmem_output[output_size - lane_id - 1] = __float2half(0);
  }
}

inline void dotBasedInteractFwd(
    const void *input, const void *bottom_mlp_output, void *output, uint batch_size, uint num_rows, uint num_cols) {
  const uint kWarpSize = 32;
  const uint kWarpSizeLog2 = Log2<kWarpSize>::value;
  const uint kTileDim = 16;
  const uint kTileDimLog2 = Log2<kTileDim>::value;
  const uint warps_per_threadblock = 4;
  const uint threadblock_size = warps_per_threadblock * 32;
  const uint kRowTilesPerStep = 2;
  const uint kColTilesPerStep = 1;

  // num tiles
  uint num_row_tiles = (num_rows + kTileDim - 1) >> kTileDimLog2;
  uint num_col_tiles = (num_cols + kTileDim - 1) >> kTileDimLog2;

  // number of rows and columns after padding
  uint num_rows_after_padding = kTileDim << 1; //32 rows
  uint num_cols_after_padding = num_col_tiles << kTileDimLog2; //num cols rounded up to 16

  uint num_row_steps = num_row_tiles / kRowTilesPerStep;
  uint num_col_steps = num_col_tiles / kColTilesPerStep;

  const uint K_BLOCKS = 8;
  const uint M_BLOCKS = 2;
  const uint SKEW_HALF = ((K_BLOCKS % 2) == 0) ? 8 : 0;
  const uint SMEM_STRIDE = (K_BLOCKS * 16 + SKEW_HALF);
  // multiple of 2 to guarantee 256-bit alignment for start of the row, at least 16 to safeload a tile
  const uint smem_rows_per_warp = M_BLOCKS << 4;
  const uint smem_elems_per_warp_mat = smem_rows_per_warp * SMEM_STRIDE;
  const uint SKEW_HALF_ACC = ((M_BLOCKS % 2) == 0) ? 8 : 0;
  const uint SMEM_STRIDE_ACC = (M_BLOCKS * 16 + SKEW_HALF_ACC);
  const uint smem_elems_per_warp_acc = M_BLOCKS * 16 * SMEM_STRIDE_ACC * 2;  // output in FP32
  const uint smem_elems_per_warp =
      (smem_elems_per_warp_mat > smem_elems_per_warp_acc) ? smem_elems_per_warp_mat : smem_elems_per_warp_acc;
  uint raw_output_size = ((num_rows * (num_rows - 1)) >> 1) + num_cols;
  uint output_size = ((raw_output_size-1)/8 + 1)*8; //round up to multiple of 8
  uint padding_size = output_size-raw_output_size;

  bool float4_predicate = !((num_cols & 7) || (output_size & 7));

  if (float4_predicate) {
    dotBasedInteractFwdKernel<warps_per_threadblock,
                              threadblock_size,
                              M_BLOCKS,
                              K_BLOCKS,
                              SMEM_STRIDE,
                              SMEM_STRIDE_ACC,
                              kWarpSize,
                              kWarpSizeLog2,
                              kTileDim,
                              kTileDimLog2>
        <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock, //each threadblock covers warps_per_threadblock samples, each warp covers a sample
           threadblock_size,
           warps_per_threadblock * smem_elems_per_warp * sizeof(__half),
           at::cuda::getCurrentCUDAStream()>>>((const __half *)input,
                                              (half *)output,
                                              batch_size,
                                              num_rows,
                                              num_cols,
                                              num_rows_after_padding,
                                              num_cols_after_padding,
                                              smem_elems_per_warp,
                                              smem_rows_per_warp,
                                              output_size,
                                              num_row_steps,
                                              num_col_steps,
                                              padding_size);
  } else {
    dotBasedInteractFwdKernelNonAligned<warps_per_threadblock,
                                        threadblock_size,
                                        M_BLOCKS,
                                        K_BLOCKS,
                                        SMEM_STRIDE,
                                        SMEM_STRIDE_ACC,
                                        kWarpSize,
                                        kWarpSizeLog2,
                                        kTileDim,
                                        kTileDimLog2>
        <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock,
           threadblock_size,
           warps_per_threadblock * smem_elems_per_warp * sizeof(__half),
           at::cuda::getCurrentCUDAStream()>>>((const __half *)input,
                                              (half *)output,
                                              batch_size,
                                              num_rows,
                                              num_cols,
                                              num_rows_after_padding,
                                              num_cols_after_padding,
                                              smem_elems_per_warp,
                                              smem_rows_per_warp,
                                              output_size,
                                              num_row_steps,
                                              num_col_steps,
                                              padding_size);
  }
}