#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>
#include <torch/torch.h>
#include <vector>
#include <iostream>

__global__ void box_encode_kernel(float *targets_dx, float *targets_dy, float *targets_dw, float *targets_dh,  
                                  float4 *boxes, float4 *anchors, float wx, float wy, float ww, float wh, 
                                  size_t gt, size_t idxJump) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row_offset; 
    float anchors_x1, anchors_x2, anchors_y1, anchors_y2, 
        boxes_x1, boxes_x2, boxes_y1, boxes_y2, ex_w, ex_h, 
        ex_ctr_x, ex_ctr_y, gt_w, gt_h, gt_ctr_x, gt_ctr_y;
          
    for (int i = idx; i < gt; i += idxJump){
        row_offset = i;
        anchors_x1 = anchors[row_offset].x;
        anchors_y1 = anchors[row_offset].y;
        anchors_x2 = anchors[row_offset].z;
        anchors_y2 = anchors[row_offset].w;        

        boxes_x1 = boxes[row_offset].x;
        boxes_y1 = boxes[row_offset].y;
        boxes_x2 = boxes[row_offset].z;
        boxes_y2 = boxes[row_offset].w; 
        
        ex_w = anchors_x2 - anchors_x1 + 1;
        ex_h = anchors_y2 - anchors_y1 + 1;
        ex_ctr_x = anchors_x1 + 0.5 * ex_w; 
        ex_ctr_y = anchors_y1 + 0.5 * ex_h;
               
        gt_w = boxes_x2 - boxes_x1 + 1;
        gt_h = boxes_y2 -  boxes_y1 + 1; 
        gt_ctr_x = boxes_x1 + 0.5 * gt_w; 
        gt_ctr_y = boxes_y1 + 0.5 * gt_h;        
        
        targets_dx[i] = wx * (gt_ctr_x - ex_ctr_x) / ex_w; 
        targets_dy[i] = wy * (gt_ctr_y - ex_ctr_y) / ex_h; 
        targets_dw[i] = ww * log(gt_w / ex_w); 
        targets_dh[i] = wh * log(gt_h / ex_h);          
    }  

}


std::vector<at::Tensor> box_encode_cuda(at::Tensor boxes, at::Tensor anchors, float wx, float wy, float ww, float wh){
   
    int minGridSize;
    int blockSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (void*) box_encode_kernel,
                                       0,  // dynamic memory
                                       0); // maximum utilized threads    
    long size = boxes.size(0);
    auto targets_dx = torch::ones({size}, torch::CUDA(at::kFloat)); 
    auto targets_dy = torch::ones({size}, torch::CUDA(at::kFloat));
    auto targets_dw = torch::ones({size}, torch::CUDA(at::kFloat));
    auto targets_dh = torch::ones({size}, torch::CUDA(at::kFloat));
    
    dim3 gridDim(minGridSize);
    dim3 blockDim(blockSize);
    int idxJump = minGridSize * blockSize;
    auto stream = at::cuda::getCurrentCUDAStream();
    box_encode_kernel<<<gridDim,blockDim,0,stream.stream()>>>(targets_dx.data_ptr<float>(), 
                                                              targets_dy.data_ptr<float>(), 
                                                              targets_dw.data_ptr<float>(), 
                                                              targets_dh.data_ptr<float>(), 
                                                              (float4*) boxes.data_ptr<float>(), 
                                                              (float4*) anchors.data_ptr<float>(), 
                                                              wx, wy, ww, wh, 
                                                              size, idxJump);
     
    std::vector<at::Tensor> result;
    result.push_back(targets_dx);
    result.push_back(targets_dy);
    result.push_back(targets_dw);
    result.push_back(targets_dh);  
    return result;
}
