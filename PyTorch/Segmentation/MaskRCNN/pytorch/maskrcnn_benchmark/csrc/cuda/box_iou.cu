#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>
#include <torch/torch.h>
#include <iostream>

__global__ void box_iou_cuda_kernel(float *box_iou, float4 *box1, float4 *box2, long M, 
                                    long N, int idxJump) {

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    size_t b1_idx, b2_idx, b1_row_offset, b2_row_offset; 
    float xmin1, xmin2, xmax1, xmax2, ymin1, ymin2, ymax1, ymax2;
    float x_tl, y_tl, x_br, y_br, w, h, inter, area1, area2, iou;
          
    for (long i = idx; i < M * N; i += idxJump){
        
        b1_idx = i / N;
        b2_idx = i % N;
        b1_row_offset = b1_idx;
        b2_row_offset = b2_idx;

        xmin1 = box1[b1_row_offset].x;
        ymin1 = box1[b1_row_offset].y;
        xmax1 = box1[b1_row_offset].z;
        ymax1 = box1[b1_row_offset].w;
        xmin2 = box2[b2_row_offset].x;
        ymin2 = box2[b2_row_offset].y;
        xmax2 = box2[b2_row_offset].z;
        ymax2 = box2[b2_row_offset].w;

        x_tl = fmaxf(xmin1, xmin2);
        y_tl = fmaxf(ymin1, ymin2);

        x_br = fminf(xmax1, xmax2);
        y_br = fminf(ymax1, ymax2);                                
        w = (x_br - x_tl + 1) < 0 ? 0.0f : (x_br - x_tl + 1);
        h = (y_br - y_tl + 1) < 0 ? 0.0f : (y_br - y_tl + 1);

        inter = w * h;
        area1 = (xmax1 - xmin1 + 1) * (ymax1 - ymin1 + 1);
        area2 = (xmax2 - xmin2 + 1) * (ymax2 - ymin2 + 1);
        iou = inter / (area1 + area2 - inter);
        box_iou[b1_idx * N + b2_idx] = iou;
    }  

}

at::Tensor box_iou_cuda(at::Tensor box1, at::Tensor box2){

    int minGridSize;
    int blockSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (void*) box_iou_cuda_kernel,
                                       0,  // dynamic memory
                                       0); // maximum utilized threads
    
    long M = box1.size(0);
    long N = box2.size(0);
    auto box_iou = torch::ones({M, N}, torch::CUDA(at::kFloat));
    
    dim3 gridDim(minGridSize);
    dim3 blockDim(blockSize);
    int idxJump = minGridSize * blockSize;
    auto stream = at::cuda::getCurrentCUDAStream();
    box_iou_cuda_kernel<<<gridDim, blockDim, 0, stream.stream()>>>(box_iou.data_ptr<float>(), 
                                                                  (float4*) box1.data_ptr<float>(), 
                                                                  (float4*) box2.data_ptr<float>(), 
                                                                  M, N, 
                                                                  idxJump);
    return box_iou;
}

