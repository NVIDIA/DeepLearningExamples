#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__device__ T bilinear_interpolate(const T* bottom_data,
    const int height, const int width,
    T y, T x,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}


template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width,
    T y, T x,
    T & w1, T & w2, T & w3, T & w4,
    int & x_low, int & x_high, int & y_low, int & y_high,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int) y;
  x_low = (int) x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename T>
__global__ void RoIAlignBackwardFeature(const int nthreads, const T* top_diff,
    const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(height, width, y, x,
            w1, w2, w3, w4,
            x_low, x_high, y_low, y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
        {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward


at::Tensor ROIAlign_forward_cuda(const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width,
                                 const int sampling_ratio) {
  AT_ASSERTM(input.is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIAlign_forward", [&] {
    RoIAlignForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         input.contiguous().data_ptr<scalar_t>(),
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         sampling_ratio,
         rois.contiguous().data_ptr<scalar_t>(),
         output.data_ptr<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return output;
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor ROIAlign_backward_cuda(const at::Tensor& grad,
                                  const at::Tensor& rois,
                                  const float spatial_scale,
                                  const int pooled_height,
                                  const int pooled_width,
                                  const int batch_size,
                                  const int channels,
                                  const int height,
                                  const int width,
                                  const int sampling_ratio) {
  AT_ASSERTM(grad.is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.scalar_type(), "ROIAlign_backward", [&] {
    RoIAlignBackwardFeature<scalar_t><<<grid, block, 0, stream>>>(
         grad.numel(),
         grad.contiguous().data_ptr<scalar_t>(),
         num_rois,
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         sampling_ratio,
         grad_input.data_ptr<scalar_t>(),
         rois.contiguous().data_ptr<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}
