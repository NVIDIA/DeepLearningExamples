#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void RoIPoolFForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const T* bottom_rois, T* top_data, int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    int roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    int roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    int roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height)
                       / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width)
                       / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (offset_bottom_data[bottom_index] > maxval) {
          maxval = offset_bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void RoIPoolFBackward(const int nthreads, const T* top_diff,
    const int* argmax_data, const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;
    const int* offset_argmax_data = argmax_data + top_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      atomicAdd(
          offset_bottom_diff + argmax,
          static_cast<T>(offset_top_diff[ph * pooled_width + pw]));

    }
  }
}

std::tuple<at::Tensor, at::Tensor> ROIPool_forward_cuda(const at::Tensor& input,
                                const at::Tensor& rois,
                                const float spatial_scale,
                                const int pooled_height,
                                const int pooled_width) {
  AT_ASSERTM(input.is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  auto argmax = at::zeros({num_rois, channels, pooled_height, pooled_width}, input.options().dtype(at::kInt));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return std::make_tuple(output, argmax);
  }

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIPool_forward", [&] {
    RoIPoolFForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         input.contiguous().data_ptr<scalar_t>(),
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         rois.contiguous().data_ptr<scalar_t>(),
         output.data_ptr<scalar_t>(),
         argmax.data_ptr<int>());
  });
  THCudaCheck(hipGetLastError());
  return std::make_tuple(output, argmax);
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor ROIPool_backward_cuda(const at::Tensor& grad,
                                 const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const at::Tensor& argmax,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width,
                                 const int batch_size,
                                 const int channels,
                                 const int height,
                                 const int width) {
  AT_ASSERTM(grad.is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");
  // TODO add more checks

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.scalar_type(), "ROIPool_backward", [&] {
    RoIPoolFBackward<scalar_t><<<grid, block, 0, stream>>>(
         grad.numel(),
         grad.contiguous().data_ptr<scalar_t>(),
         argmax.data_ptr<int>(),
         num_rois,
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         grad_input.data_ptr<scalar_t>(),
         rois.contiguous().data_ptr<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}
